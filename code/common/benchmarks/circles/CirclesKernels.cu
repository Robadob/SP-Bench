#include "hip/hip_runtime.h"
#include "CirclesKernels.cuh"

__device__ __constant__ float d_attract;
__device__ __constant__ float d_repulse;

__global__ void init_curand(hiprandState *state, unsigned long long seed) {
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    if (id < d_locationMessageCount)
        hiprand_init(seed, id, 0, &state[id]);
}
__global__ void init_particles(hiprandState *state, LocationMessages *locationMessages) {
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    if (id >= d_locationMessageCount)
        return;
    //curand_unform returns 0<x<=1.0, not much can really do about 0 exclusive
    //negate and  + 1.0, to make  0<=x<1.0
    locationMessages->locationX[id] = (-hiprand_uniform(&state[id]) + 1.0f)*d_environmentMax.x;
    locationMessages->locationY[id] = (-hiprand_uniform(&state[id]) + 1.0f)*d_environmentMax.y;
#ifdef _3D
    locationMessages->locationZ[id] = (-hiprand_uniform(&state[id]) + 1.0f)*d_environmentMax.z;
#endif
}
__global__ void init_particles_uniform(LocationMessages *locationMessages) {
	int id = blockIdx.x * blockDim.x + threadIdx.x;
	if (id >= d_locationMessageCount)
		return;
	int hash = id % (glm::compMul(d_gridDim));
	int div = id / (glm::compMul(d_gridDim));
	int max = d_locationMessageCount / (glm::compMul(d_gridDim));
	int z = (hash / (d_gridDim.y * d_gridDim.x));
	int y = (hash % (d_gridDim.y * d_gridDim.x)) / d_gridDim.x;
	int x = (hash % (d_gridDim.y * d_gridDim.x)) % d_gridDim.x;
	//In a regular manner, scatter particles evenly between bins
	locationMessages->locationX[id] = (x * (d_environmentMax.x / (float)d_gridDim.x)) + (d_environmentMax.x / (float)d_gridDim.x)*0.5;
	locationMessages->locationY[id] = (y * (d_environmentMax.y / (float)d_gridDim.y)) + (d_environmentMax.y / (float)d_gridDim.y)*0.5;
#ifdef _3D
	locationMessages->locationZ[id] = (z * (d_environmentMax.z / (float)d_gridDim.z)) + (d_environmentMax.z / (float)d_gridDim.z)*0.5;
#endif
}
__global__ void step_model(LocationMessages *locationMessagesIn, LocationMessages *locationMessagesOut)
{

    int id = blockIdx.x * blockDim.x + threadIdx.x;
    if (id >= d_locationMessageCount)
        return;

    //Get my local location
#ifdef _3D
    glm::vec3 myLoc(locationMessagesIn->locationX[id], locationMessagesIn->locationY[id], locationMessagesIn->locationZ[id]), locDiff, newLoc;
#else
	glm::vec2 myLoc(locationMessagesIn->locationX[id], locationMessagesIn->locationY[id]), locDiff, newLoc;
#endif
	newLoc = myLoc;
	//Get first message
    float dist, separation, k;
    LocationMessage *lm = locationMessagesIn->getFirstNeighbour(myLoc);
    //Always atleast 1 location message, our own location!
    do
    {
        if ((lm->id != id))
        {
			locDiff = myLoc - lm->location;//Difference
            if (locDiff!=DIMENSIONS_VEC(0))//Ignore distance 0
			{
				dist = length(locDiff);//Distance (via pythagoras)
				separation = dist - d_interactionRad;
				if (separation < d_interactionRad)
				{

					k = (separation > 0.0f) ? d_attract : -d_repulse;
					newLoc += (k * separation * locDiff / d_interactionRad);
				}
            }
        }
        lm = locationMessagesIn->getNextNeighbour(lm);//Returns a pointer to shared memory or 0
    } while (lm);
    //Export newLoc
	newLoc = glm::clamp(newLoc, d_environmentMin, d_environmentMax);
	locationMessagesOut->locationX[id] = newLoc.x;
	locationMessagesOut->locationY[id] = newLoc.y;
#ifdef _3D
	locationMessagesOut->locationZ[id] = newLoc.z;
#endif
}