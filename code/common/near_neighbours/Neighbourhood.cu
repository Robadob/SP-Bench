#include "hip/hip_runtime.h"
#include "NeighbourhoodConstants.cuh"

#if defined(MORTON)
#include "Morton.h"
#elif defined(HILBERT)
#include "Hilbert.h"
#elif defined(PEANO)
#include "Peano.h"
#elif defined(MORTON_COMPUTE)
#include "MortonCompute.h"
#endif

#include "Neighbourhood.cuh"
#include "NeighbourhoodKernels.cuh"

#ifndef THRUST
#include <hipcub/hipcub.hpp>
#else
#include <thrust/sort.h>
#include <thrust/system/cuda/execution_policy.h>
#endif
#ifdef _GL
#include <cuda_gl_interop.h>
#endif
#ifdef _DEBUG
#include <glm/gtc/epsilon.hpp>
#endif
SpatialPartition::SpatialPartition(DIMENSIONS_VEC  environmentMin, DIMENSIONS_VEC environmentMax, unsigned int maxAgents, float interactionRad)
    : maxAgents(maxAgents)
    , interactionRad(interactionRad)
    , locationMessageCount(0)
    , environmentMin(environmentMin)
    , environmentMax(environmentMax)
    , gridDim((environmentMax - environmentMin) / interactionRad)
#if defined(MORTON) || defined(HILBERT) || defined(PEANO) || defined(MORTON_COMPUTE)
    , gridExponent(0)
#endif
#ifdef _DEBUG
    , PBM_isBuilt(0)
#endif
{
#ifdef _DEBUG
//#if defined(_2D)
//    printf("Spatial Partition: Interaction Rad(%.3f), Grid Dims(%d,%d)\n", interactionRad, gridDim.x, gridDim.y);
//#elif defined(_3D)
//    printf("Spatial Partition: Interaction Rad(%.3f), Grid Dims(%d,%d,%d)\n", interactionRad, gridDim.x, gridDim.y, gridDim.z);
//#endif
#endif
    setBinCount();
    //Allocate bins in GPU memory
    deviceAllocateLocationMessages(&d_locationMessages, &hd_locationMessages);
    //Allocate bins swap in GPU memory
    deviceAllocateLocationMessages(&d_locationMessages_swap, &hd_locationMessages_swap);
    //Allocate PBM
    deviceAllocatePBM(&d_PBM);
    //Allocate primitive structures
    deviceAllocatePrimitives(&d_keys, &d_vals);
#ifndef THRUST
    deviceAllocatePrimitives(&d_keys_swap, &d_vals_swap);
    deviceAllocateCUBTemp(&d_CUB_temp_storage, d_CUB_temp_storage_bytes);
#endif
    //Allocate tex
    deviceAllocateTextures();
    //Set device constants
    CUDA_CALL(hipMemcpyToSymbol(HIP_SYMBOL(d_interactionRad), &interactionRad, sizeof(float)));
    CUDA_CALL(hipMemcpyToSymbol(HIP_SYMBOL(d_gridDim), &gridDim, sizeof(DIMENSIONS_IVEC)));
    DIMENSIONS_VEC t_gridDim = (DIMENSIONS_VEC)gridDim;
    CUDA_CALL(hipMemcpyToSymbol(HIP_SYMBOL(d_gridDim_float), &t_gridDim, sizeof(DIMENSIONS_VEC)));

    CUDA_CALL(hipMemcpyToSymbol(HIP_SYMBOL(d_environmentMin), &environmentMin, sizeof(DIMENSIONS_VEC)));
    CUDA_CALL(hipMemcpyToSymbol(HIP_SYMBOL(d_environmentMax), &environmentMax, sizeof(DIMENSIONS_VEC)));

#ifdef _DEBUG
    CUDA_CALL(hipMemcpyToSymbol(HIP_SYMBOL(d_PBM_isBuilt), &PBM_isBuilt, sizeof(unsigned int)));
#endif
    setLocationCount(locationMessageCount);
    unsigned int t_binCount = this->binCountMax;
    CUDA_CALL(hipMemcpyToSymbol(HIP_SYMBOL(d_binCount), &t_binCount, sizeof(unsigned int)));

#if defined(_GL) || defined(_DEBUG)
    CUDA_CALL(hipMemcpyToSymbol(HIP_SYMBOL(d_locationMessagesA), &d_locationMessages, sizeof(LocationMessages *)));
    CUDA_CALL(hipMemcpyToSymbol(HIP_SYMBOL(d_locationMessagesB), &d_locationMessages_swap, sizeof(LocationMessages *)));
#endif
    //Init lookup table
#if defined(MORTON)
    initMorton(gridDim);
#elif defined(HILBERT)
    initHilbert(gridDim);
#elif defined(PEANO)
    initPeano(gridDim);
#endif
}
SpatialPartition::~SpatialPartition()
{
    CUDA_CHECK();
    //Dellocate bins in GPU memory
    deviceDeallocateLocationMessages(d_locationMessages, hd_locationMessages);
    //Dellocate bins swap in GPU memory
    deviceDeallocateLocationMessages(d_locationMessages_swap, hd_locationMessages_swap);
    //Dellocate PBM
    deviceDeallocatePBM(d_PBM);
    //Deallocated primitive structures
    deviceDeallocatePrimitives(d_keys, d_vals);
#ifndef THRUST
    deviceDeallocatePrimitives(d_keys_swap, d_vals_swap);
    deviceDeallocateCUBTemp(d_CUB_temp_storage);
#endif
    //Deallocate tex
    deviceDeallocateTextures();
    //Free lookup table
#if defined(MORTON)
    freeMorton();
#elif defined(HILBERT)
    freeHilbert();
#elif defined(PEANO)
    freePeano();
#endif
}
#ifdef _DEBUG

//DIMENSIONS_IVEC SpatialPartition::getGridPosition(DIMENSIONS_VEC worldPos)
//{
//#ifndef SP_NO_CLAMP_GRID
//    //Clamp each grid coord to 0<=x<dim
//    return clamp(floor(((worldPos - environmentMin) / (environmentMax - environmentMin))*glm::vec3(gridDim)), glm::vec3(0), glm::vec3(gridDim)-glm::vec3(1));
//#else
//    return floor(((worldPos - environmentMin) / (environmentMax - environmentMin))*glm::vec3(gridDim));
//#endif
//}
//
//unsigned int SpatialPartition::getHash(DIMENSIONS_IVEC gridPos)
//{
//    gridPos = clamp(gridPos, DIMENSIONS_IVEC(0), gridDim - DIMENSIONS_IVEC(1));
//    return
//#ifdef _3D
//        (gridPos.z * gridDim.y * gridDim.x) +   //z
//#endif
//        (gridPos.y * gridDim.x) +					//y
//        gridPos.x; 	                                //x
//}
int SpatialPartition::getHash(DIMENSIONS_IVEC gridPos)
{//Host version using host copy of gridDim
    gridPos = glm::clamp(gridPos, DIMENSIONS_IVEC(0), gridDim - DIMENSIONS_IVEC(1));
#if defined(MORTON)
    return h_mortonEncode(gridPos);
#elif defined(HILBERT)
    return h_hilbertEncode(gridPos);
#elif defined(PEANO)
    return h_peanoEncode(gridPos);
#elif defined(MORTON_COMPUTE)
    return mortonComputeEncode(gridPos);
#else
    return
#ifdef _3D
        (gridPos.z * gridDim.y * gridDim.x) +   //z
#endif
        (gridPos.y * gridDim.x) +					//y
        gridPos.x;
#endif
}
DIMENSIONS_IVEC SpatialPartition::getPos(unsigned int hash)
{
#if defined(MORTON)
    return mortonDecode(hash);
#elif defined(HILBERT)
    return hilbertDecode(hash, this->gridExponent);
#elif defined(PEANO)
    return peanoDecode(hash, this->gridExponent);
#elif defined(MORTON_COMPUTE)
    return mortonDecode(hash);
#else
    if (hash >= this->binCountMax)
        return DIMENSIONS_IVEC(-1);
    else
    {
#ifdef _3D

        int z = (hash / (gridDim.y * gridDim.x));
        int y = (hash % (gridDim.y * gridDim.x)) / gridDim.x;
        int x = (hash % (gridDim.y * gridDim.x)) % gridDim.x;
        return DIMENSIONS_IVEC(x, y, z);
#else
        int y = hash / gridDim.x;
        int x = hash % gridDim.x;
        return DIMENSIONS_IVEC(x, y);
#endif
    }
#endif
}
bool SpatialPartition::isValid(DIMENSIONS_IVEC bin) const
{
    if (
#ifdef _3D
        bin.z<0 || bin.z >= gridDim.z ||
#endif
        bin.y<0 || bin.y >= gridDim.y ||
        bin.x<0 || bin.x >= gridDim.x
        )
    {
        return false;
    }
    return true;
}
void SpatialPartition::assertSearch()
{
    //return;//
    unsigned int outCount = this->binCountMax + 1;
    unsigned int tableSize = ((outCount / 10) + 1) * 10;

    //Copy raw PBM from device to host
    unsigned int *PBM_raw = static_cast<unsigned int *>(malloc(sizeof(unsigned int)*tableSize));
    memset(PBM_raw, 0, tableSize * sizeof(unsigned int));
    CUDA_CALL(hipMemcpy(PBM_raw, d_PBM, sizeof(unsigned int)*outCount, hipMemcpyDeviceToHost));

    //Calculate the size of every bin
    unsigned int agtCount = 0;
    unsigned int *PBM_binSize = static_cast<unsigned int *>(malloc(sizeof(unsigned int)*tableSize));
    for (unsigned int i = 0; i < tableSize; i++)
    {
        if (i < outCount - 1)
        {
            PBM_binSize[i] = PBM_raw[i + 1] - PBM_raw[i];
            agtCount += PBM_binSize[i];
        }
        else
        {
            PBM_binSize[i] = 11111;
        }

    }
    if (agtCount != maxAgents&&agtCount != 0)
    {
        printf("%i PBM records exist for %i agents.\n", agtCount, maxAgents);
    }

#if defined(MORTON) || defined(HILBERT) || defined(PEANO) || defined(MORTON_COMPUTE)
    //In the case of morton coding, we sort PBM back into our regular order to compare
    unsigned int *PBM_coded = PBM_binSize;
    PBM_binSize = static_cast<unsigned int *>(malloc(sizeof(unsigned int)*tableSize));
    memset(PBM_binSize, 0, tableSize * sizeof(unsigned int));
    for (int i = 0; i < this->binCount; i++)
    {
#if defined(_3D)
        PBM_binSize[i] = PBM_coded[getHash(glm::ivec3((i % (gridDim.y * gridDim.x)) % gridDim.x, (i % (gridDim.y * gridDim.x)) / gridDim.x, (i / (gridDim.y * gridDim.x))))];
#elif defined(_2D)
        PBM_binSize[i] = PBM_coded[getHash(glm::ivec2(i % gridDim.x, i / gridDim.x))];
#endif
    }
    free(PBM_coded);
#endif
    //Calculate the size of each bin's neighbourhood
    unsigned int *PBM_neighbourhoodSize = static_cast<unsigned int *>(malloc(sizeof(unsigned int)*tableSize));
    for (unsigned int i = 0; i < ((outCount / 10) + 1) * 10; i++)
    {
        PBM_neighbourhoodSize[i] = 0;
        if (i < outCount - 1)
        {
            DIMENSIONS_IVEC curCell = getPos(i);
            for (int x = -1; x <= 1; x++)
                for (int y = -1; y <= 1; y++)
                {
#if defined(_2D)
                DIMENSIONS_IVEC neighbourCell = curCell + DIMENSIONS_IVEC(x, y);
#elif defined(_3D)
                for (int z = -1; z <= 1; z++)
                {
                    DIMENSIONS_IVEC neighbourCell = curCell + DIMENSIONS_IVEC(x, y, z);
#endif
                    if (isValid(neighbourCell))
                    {
                        PBM_neighbourhoodSize[i] += PBM_binSize[getHash(neighbourCell)];
                    }
#ifdef _3D
                }
#endif
                }
        }

    }

    //Copy every location and neighbour count from device to host
    float *d_bufferPtr;
    LocationMessages lm;
    lm.locationX = (float*)malloc(sizeof(float)*locationMessageCount);
    CUDA_CALL(hipMemcpy(&d_bufferPtr, &d_locationMessages_swap->locationX, sizeof(float*), hipMemcpyDeviceToHost));
    CUDA_CALL(hipMemcpy(lm.locationX, d_bufferPtr, sizeof(float)*locationMessageCount, hipMemcpyDeviceToHost));
    lm.locationY = (float*)malloc(sizeof(float)*locationMessageCount);
    CUDA_CALL(hipMemcpy(&d_bufferPtr, &d_locationMessages_swap->locationY, sizeof(float*), hipMemcpyDeviceToHost));
    CUDA_CALL(hipMemcpy(lm.locationY, d_bufferPtr, sizeof(float)*locationMessageCount, hipMemcpyDeviceToHost));
#ifdef _3D
    lm.locationZ = (float*)malloc(sizeof(float)*locationMessageCount);
    CUDA_CALL(hipMemcpy(&d_bufferPtr, &d_locationMessages_swap->locationZ, sizeof(float*), hipMemcpyDeviceToHost));
    CUDA_CALL(hipMemcpy(lm.locationZ, d_bufferPtr, sizeof(float)*locationMessageCount, hipMemcpyDeviceToHost));
#endif
    lm.count = (float*)malloc(sizeof(float)*locationMessageCount);
    CUDA_CALL(hipMemcpy(&d_bufferPtr, &d_locationMessages_swap->count, sizeof(float*), hipMemcpyDeviceToHost));
    CUDA_CALL(hipMemcpy(lm.count, d_bufferPtr, sizeof(float)*locationMessageCount, hipMemcpyDeviceToHost));
    //ASSERT: Every agent searched the right amount of neighbours
    unsigned int matchFails = 0;
    for (unsigned int i = 0; i < locationMessageCount; i++)
    {
        //For rendering purposes the count is stored as count/totalMessages, invert this math for assertion
#if defined(_2D)
        unsigned int hash = getHash(getGridPosition(DIMENSIONS_VEC(lm.locationX[i], lm.locationY[i])));
#elif defined(_3D)
        unsigned int hash = getHash(getGridPosition(DIMENSIONS_VEC(lm.locationX[i], lm.locationY[i], lm.locationZ[i])));
#endif
        if (glm::epsilonNotEqual(lm.count[i], PBM_neighbourhoodSize[hash] / (float)locationMessageCount, 0.5f))
        {
            //printf("%u=%u-%f=%f,", (unsigned int)(lm.count[i] * locationMessageCount), PBM_neighbourhoodSize[hash], lm.count[i], PBM_neighbourhoodSize[hash] / (float)locationMessageCount);
            matchFails++;
        }
    }
    //Free location/count data
    free(lm.locationX);
    free(lm.locationY);
#ifdef _3D
    free(lm.locationZ);
#endif
    free(lm.count);
    if (matchFails>0)
    {
        printf("ERROR: Neighbour search totals do not match (%u/%u)\n", matchFails, locationMessageCount);
    }
    else
    {
        free(PBM_raw);
        free(PBM_binSize);
        free(PBM_neighbourhoodSize);
        return;
    }
    //Output the 3 PBM_ data structures to file in a readable format
    FILE *file = fopen("../logs/PBM.txt", "w");
    fprintf(file, "ERROR: Neighbour search totals do not match (%u/%u)\n", matchFails, locationMessageCount);
    fprintf(file, "Raw PBM\n");
    fprintf(file, "|%5s|%5s|%5s|%5s|%5s|%5s|%5s|%5s|%5s|%5s|%5s|\n", "", "0", "1", "2", "3", "4", "5", "6", "7", "8", "9");
    fprintf(file, "|-----|-----|-----|-----|-----|-----|-----|-----|-----|-----|-----|\n");
    for (unsigned int i = 0; i < (outCount / 10) - 1; i++)
    {
        fprintf(file, "|%4u0|%5u|%5u|%5u|%5u|%5u|%5u|%5u|%5u|%5u|%5u|\n", i,
            PBM_raw[(10 * i) + 0],
            PBM_raw[(10 * i) + 1],
            PBM_raw[(10 * i) + 2],
            PBM_raw[(10 * i) + 3],
            PBM_raw[(10 * i) + 4],
            PBM_raw[(10 * i) + 5],
            PBM_raw[(10 * i) + 6],
            PBM_raw[(10 * i) + 7],
            PBM_raw[(10 * i) + 8],
            PBM_raw[(10 * i) + 9]
            );
    }
    fprintf(file, "Bin Size\n");
    fprintf(file, "|%5s|%5s|%5s|%5s|%5s|%5s|%5s|%5s|%5s|%5s|%5s|\n", "", "0", "1", "2", "3", "4", "5", "6", "7", "8", "9");
    fprintf(file, "|-----|-----|-----|-----|-----|-----|-----|-----|-----|-----|-----|\n");
    for (unsigned int i = 0; i < ((this->binCount + 1) / 10) - 1; i++)
    {
        fprintf(file, "|%4u0|%5u|%5u|%5u|%5u|%5u|%5u|%5u|%5u|%5u|%5u|\n", i,
            PBM_binSize[(10 * i) + 0],
            PBM_binSize[(10 * i) + 1],
            PBM_binSize[(10 * i) + 2],
            PBM_binSize[(10 * i) + 3],
            PBM_binSize[(10 * i) + 4],
            PBM_binSize[(10 * i) + 5],
            PBM_binSize[(10 * i) + 6],
            PBM_binSize[(10 * i) + 7],
            PBM_binSize[(10 * i) + 8],
            PBM_binSize[(10 * i) + 9]
            );
    }
    fprintf(file, "Neighbour Count\n");
    fprintf(file, "|%5s|%5s|%5s|%5s|%5s|%5s|%5s|%5s|%5s|%5s|%5s|\n", "", "0", "1", "2", "3", "4", "5", "6", "7", "8", "9");
    fprintf(file, "|-----|-----|-----|-----|-----|-----|-----|-----|-----|-----|-----|\n");
    for (unsigned int i = 0; i < (outCount / 10) - 1; i++)
    {
        fprintf(file, "|%4u0|%5u|%5u|%5u|%5u|%5u|%5u|%5u|%5u|%5u|%5u|\n", i,
            PBM_neighbourhoodSize[(10 * i) + 0],
            PBM_neighbourhoodSize[(10 * i) + 1],
            PBM_neighbourhoodSize[(10 * i) + 2],
            PBM_neighbourhoodSize[(10 * i) + 3],
            PBM_neighbourhoodSize[(10 * i) + 4],
            PBM_neighbourhoodSize[(10 * i) + 5],
            PBM_neighbourhoodSize[(10 * i) + 6],
            PBM_neighbourhoodSize[(10 * i) + 7],
            PBM_neighbourhoodSize[(10 * i) + 8],
            PBM_neighbourhoodSize[(10 * i) + 9]
            );
    }
    //Cleanup resources
    fclose(file);
    free(PBM_raw);
    free(PBM_binSize);
    free(PBM_neighbourhoodSize);
}
#endif
void SpatialPartition::deviceAllocateLocationMessages(LocationMessages **d_locMessage, LocationMessages *hd_locMessage)
{
    CUDA_CALL(hipMalloc(d_locMessage, sizeof(LocationMessages)));
    CUDA_CALL(hipMalloc(&hd_locMessage->locationX, sizeof(float)*maxAgents));
    CUDA_CALL(hipMemcpy(&((*d_locMessage)->locationX), &(hd_locMessage->locationX), sizeof(float*), hipMemcpyHostToDevice));
    CUDA_CALL(hipMalloc(&hd_locMessage->locationY, sizeof(float)*maxAgents));
    CUDA_CALL(hipMemcpy(&((*d_locMessage)->locationY), &(hd_locMessage->locationY), sizeof(float*), hipMemcpyHostToDevice));
#ifdef _3D
    CUDA_CALL(hipMalloc(&hd_locMessage->locationZ, sizeof(float)*maxAgents));
    CUDA_CALL(hipMemcpy(&((*d_locMessage)->locationZ), &(hd_locMessage->locationZ), sizeof(float*), hipMemcpyHostToDevice));
#endif
#if defined(_GL) || defined(_DEBUG)
    CUDA_CALL(hipMalloc(&hd_locMessage->count, sizeof(float)*maxAgents));
    CUDA_CALL(hipMemcpy(&((*d_locMessage)->count), &(hd_locMessage->count), sizeof(float*), hipMemcpyHostToDevice));
    CUDA_CALL(hipMemset(hd_locMessage->count, 0, sizeof(float)*maxAgents));//Must be 0'd to protect assertions on k20
#endif
}
void SpatialPartition::deviceAllocatePBM(unsigned int **d_PBM_t)
{
    CUDA_CALL(hipMalloc(d_PBM_t, sizeof(unsigned int)*(this->binCountMax + 1)));
    CUDA_CALL(hipMemset(*d_PBM_t, 0, sizeof(unsigned int)*(this->binCountMax + 1)));//Must be 0'd to protect assertions on k20
}
void SpatialPartition::deviceAllocatePrimitives(unsigned int **d_keys, unsigned int **d_vals)
{
    CUDA_CALL(hipMalloc(d_keys, sizeof(unsigned int)*maxAgents));
    CUDA_CALL(hipMalloc(d_vals, sizeof(unsigned int)*maxAgents));
}
#ifndef THRUST
void SpatialPartition::deviceAllocateCUBTemp(void **d_CUB_temp, size_t &d_cub_temp_bytes)
{
    //CUB version
    // Determine temporary device storage requirements
    d_cub_temp_bytes = 0;
    *d_CUB_temp = NULL;
    hipcub::DeviceRadixSort::SortPairs(*d_CUB_temp, d_cub_temp_bytes, d_keys, d_keys_swap, d_vals, d_vals_swap, maxAgents);
    // Allocate temporary storage
    CUDA_CALL(hipMalloc(d_CUB_temp, d_cub_temp_bytes));
}
#endif
void SpatialPartition::deviceAllocateTextures()
{
    //Locations
#ifdef _GL
#pragma unroll 3
    for (unsigned int i = 0; i < DIMENSIONS; i++)
        deviceAllocateGLTexture_float(i);
    deviceAllocateGLTexture_float2();//Allocate a texture to store counting info in (Used to colour the visualisation
#else
#pragma unroll 3
    for (unsigned int i = 0; i < DIMENSIONS; i++)
        deviceAllocateTexture_float(i);
#endif
    //PBM
    deviceAllocateTexture_int();
}
void SpatialPartition::fillTextures()
{
    CUDA_CALL(hipMemcpy(tex_loc_ptr[0], hd_locationMessages.locationX, locationMessageCount*sizeof(float), hipMemcpyDeviceToDevice));
    CUDA_CALL(hipMemcpy(tex_loc_ptr[1], hd_locationMessages.locationY, locationMessageCount*sizeof(float), hipMemcpyDeviceToDevice));
#ifdef _3D
    CUDA_CALL(hipMemcpy(tex_loc_ptr[2], hd_locationMessages.locationZ, locationMessageCount*sizeof(float), hipMemcpyDeviceToDevice));
#endif
#ifdef _GL
    CUDA_CALL(hipMemcpy(tex_location_ptr_count, hd_locationMessages.count, locationMessageCount*sizeof(float), hipMemcpyDeviceToDevice));
#endif
    CUDA_CALL(hipMemcpy(tex_PBM_ptr, d_PBM, (this->binCountMax + 1)*sizeof(unsigned int), hipMemcpyDeviceToDevice));
}

void SpatialPartition::deviceAllocateTexture_float(unsigned int i)
{
    if (i >= DIMENSIONS)
        return;
    //Allocate cuda array
    CUDA_CALL(hipMalloc(&tex_loc_ptr[i], maxAgents*sizeof(float)));
    //Define cuda resource from array
    hipResourceDesc resDesc;
    memset(&resDesc, 0, sizeof(hipResourceDesc));
    resDesc.resType = hipResourceTypeLinear;
    resDesc.res.linear.devPtr = tex_loc_ptr[i];
    resDesc.res.linear.desc.f = hipChannelFormatKindFloat;
    resDesc.res.linear.desc.x = 32; // bits per channel
    resDesc.res.linear.sizeInBytes = maxAgents*sizeof(float);
    //Define a cuda texture format
    hipTextureDesc texDesc;
    memset(&texDesc, 0, sizeof(hipTextureDesc));
    texDesc.readMode = hipReadModeElementType;
    //Create texture obj
    CUDA_CALL(hipCreateTextureObject(&tex_location[i], &resDesc, &texDesc, NULL));
    //Copy obj to const memory
    CUDA_CALL(hipMemcpyToSymbol(HIP_SYMBOL(d_tex_location), &tex_location[i], sizeof(hipTextureObject_t), i*sizeof(hipTextureObject_t)));
}
#ifdef _GL
void SpatialPartition::deviceAllocateGLTexture_float(unsigned int i)//GLuint *glTex, GLuint *glTbo, hipGraphicsResource_t *cuGres, hipArray_t *cuArr, hipTextureObject_t *tex, hipTextureObject_t *d_const, const unsigned int size)
{
    if (i >= DIMENSIONS)
        return;
    float *data = new float[maxAgents];
    //Gen tex
    GL_CALL(glGenTextures(1, &gl_tex[i]));
    //Gen buffer
    GL_CALL(glGenBuffers(1, &gl_tbo[i]));
    //Size buffer and tie to tex
    GL_CALL(glBindBuffer(GL_TEXTURE_BUFFER, gl_tbo[i]));
    GL_CALL(glBindBuffer(GL_TEXTURE_BUFFER, gl_tbo[i]));
    GL_CALL(glBufferData(GL_TEXTURE_BUFFER, maxAgents*sizeof(float), 0, GL_STATIC_DRAW));

    GL_CALL(glBindTexture(GL_TEXTURE_BUFFER, gl_tex[i]));
    //glBindTexture(GL_TEXTURE_2D, 0);
    GL_CALL(glTexBuffer(GL_TEXTURE_BUFFER, GL_R32F, gl_tbo[i]));
    GL_CALL(glBindBuffer(GL_TEXTURE_BUFFER, 0));
    GL_CALL(glBindTexture(GL_TEXTURE_BUFFER, 0));

    //Get CUDA handle to texture
    memset(&gl_gRes[i], 0, sizeof(hipGraphicsResource_t));
    CUDA_CALL(hipGraphicsGLRegisterBuffer(&gl_gRes[i], gl_tbo[i], cudaGraphicsMapFlagsNone));//GL_TEXTURE_BUFFER IS UNDOCUMENTED
    //Map/convert this to something hipArray_t
    CUDA_CALL(hipGraphicsMapResources(1, &gl_gRes[i]));
    CUDA_CALL(hipGraphicsResourceGetMappedPointer((void**)&tex_loc_ptr[i], 0, gl_gRes[i]));
    CUDA_CALL(hipGraphicsUnmapResources(1, &gl_gRes[i], 0));
    //Create a texture object from the hipArray_t
    hipResourceDesc resDesc;
    memset(&resDesc, 0, sizeof(hipResourceDesc));
    resDesc.resType = hipResourceTypeLinear;
    resDesc.res.linear.devPtr = tex_loc_ptr[i];
    resDesc.res.linear.desc.f = hipChannelFormatKindFloat;
    resDesc.res.linear.desc.x = 32; // bits per channel
    resDesc.res.linear.sizeInBytes = maxAgents*sizeof(float);
    hipTextureDesc texDesc;
    memset(&texDesc, 0, sizeof(hipTextureDesc));
    texDesc.readMode = hipReadModeElementType;
    CUDA_CALL(hipCreateTextureObject(&tex_location[i], &resDesc, &texDesc, nullptr));
    //Copy texture object to device constant
    CUDA_CALL(hipMemcpyToSymbol(HIP_SYMBOL(d_tex_location), &tex_location[i], sizeof(hipTextureObject_t), i*sizeof(hipTextureObject_t)));
    delete data;
}
#endif
/*
Allocates the PBM texture, which is only accessed via CUDA & memcpy
*/
void SpatialPartition::deviceAllocateTexture_int()
{
    //Define cuda array format
    //Allocate cuda array
    unsigned int size = this->binCountMax + 1;
    CUDA_CALL(hipMalloc(&tex_PBM_ptr, size*sizeof(unsigned int)));//Number of elements, not bytes
    //Define cuda resource from array
    hipResourceDesc resDesc;
    memset(&resDesc, 0, sizeof(hipResourceDesc));
    resDesc.resType = hipResourceTypeLinear;
    resDesc.res.linear.devPtr = tex_PBM_ptr;
    resDesc.res.linear.desc.f = hipChannelFormatKindUnsigned;
    resDesc.res.linear.desc.x = 32; // bits per channel
    resDesc.res.linear.sizeInBytes = size*sizeof(unsigned int);

    hipTextureDesc texDesc;
    memset(&texDesc, 0, sizeof(hipTextureDesc));
    texDesc.readMode = hipReadModeElementType;

    CUDA_CALL(hipCreateTextureObject(&tex_PBM, &resDesc, &texDesc, NULL));
    CUDA_CALL(hipMemcpyToSymbol(HIP_SYMBOL(d_tex_PBM), &tex_PBM, sizeof(hipTextureObject_t)));
}
#ifdef _GL
/*
Allocates the count texture, which is only accessed via memcpy & GL
*/
void SpatialPartition::deviceAllocateGLTexture_float2()
{
    int *data = new int[maxAgents];
    //Gen tex
    GL_CALL(glGenTextures(1, &gl_tex_count));
    //Gen buffer
    GL_CALL(glGenBuffers(1, &gl_tbo_count));
    //Size buffer and tie to tex
    GL_CALL(glBindBuffer(GL_TEXTURE_BUFFER, gl_tbo_count));
    GL_CALL(glBindBuffer(GL_TEXTURE_BUFFER, gl_tbo_count));
    GL_CALL(glBufferData(GL_TEXTURE_BUFFER, maxAgents*sizeof(float), 0, GL_STATIC_DRAW));

    GL_CALL(glBindTexture(GL_TEXTURE_BUFFER, gl_tex_count));
    //glBindTexture(GL_TEXTURE_2D, 0);
    GL_CALL(glTexBuffer(GL_TEXTURE_BUFFER, GL_R32F, gl_tbo_count));
    GL_CALL(glBindBuffer(GL_TEXTURE_BUFFER, 0));
    GL_CALL(glBindTexture(GL_TEXTURE_BUFFER, 0));

    //Get CUDA handle to texture
    memset(&gl_gRes_count, 0, sizeof(hipGraphicsResource_t));
    CUDA_CALL(hipGraphicsGLRegisterBuffer(&gl_gRes_count, gl_tbo_count, cudaGraphicsMapFlagsNone));
    //Map/convert this to something hipArray_t
    CUDA_CALL(hipGraphicsMapResources(1, &gl_gRes_count));
    CUDA_CALL(hipGraphicsResourceGetMappedPointer((void**)&tex_location_ptr_count, 0, gl_gRes_count));
    CUDA_CALL(hipGraphicsUnmapResources(1, &gl_gRes_count, 0));
    delete data;
}
#endif
void SpatialPartition::deviceDeallocateLocationMessages(LocationMessages *d_locMessage, LocationMessages hd_locMessage)
{
    CUDA_CALL(hipFree(hd_locMessage.locationX));
    CUDA_CALL(hipFree(hd_locMessage.locationY));
#ifdef _3D
    CUDA_CALL(hipFree(hd_locMessage.locationZ));
#endif
#ifdef _GL
    CUDA_CALL(hipFree(hd_locMessage.count));
#endif
    CUDA_CALL(hipFree(d_locMessage));
}
void SpatialPartition::deviceDeallocatePBM(unsigned int *d_PBM_t)
{
    CUDA_CALL(hipFree(d_PBM_t));
}
void SpatialPartition::deviceDeallocatePrimitives(unsigned int *d_keys, unsigned int *d_vals)
{
    CUDA_CALL(hipFree(d_keys));
    CUDA_CALL(hipFree(d_vals));
}
#ifndef THRUST
void SpatialPartition::deviceDeallocateCUBTemp(void *d_CUB_temp)
{
    CUDA_CALL(hipFree(d_CUB_temp));
}
#endif
void SpatialPartition::deviceDeallocateTextures()
{

#pragma unroll
    for (unsigned int i = 0; i < DIMENSIONS; i++)
    {
        hipDestroyTextureObject(tex_location[i]);
#ifdef _GL
        hipGraphicsUnregisterResource(gl_gRes[i]);
        GL_CALL(glDeleteBuffers(1, &gl_tbo[i]));
        GL_CALL(glDeleteTextures(1, &gl_tex[i]));
#else
        hipFree(tex_loc_ptr[i]);
#endif
    }
    hipDestroyTextureObject(tex_PBM);
    hipFree(tex_PBM_ptr);
#ifdef _GL
    hipGraphicsUnregisterResource(gl_gRes_count);
    GL_CALL(glDeleteBuffers(1, &gl_tbo_count));
    GL_CALL(glDeleteTextures(1, &gl_tex_count));
#endif
}

unsigned int SpatialPartition::getBinCount() const
{
    return binCountMax;
}

void SpatialPartition::setBinCount()
{
    //Get max grid dimension
    this->binCount = glm::compMax(gridDim);
    //Find the next biggest power of two
#if defined(MORTON) || defined(HILBERT) || defined(MORTON_COMPUTE)
    this->gridExponent = ceil(log2f(this->binCount));
    int l2 = pow(2, this->gridExponent);
    this->binCountMax = (unsigned int)pow(l2, DIMENSIONS);
#elif defined(PEANO)
    this->gridExponent =ceil(log(this->binCount) / log(3));
    int l3 = pow(3, this->gridExponent);
    this->binCountMax = (unsigned int)pow(l3, DIMENSIONS);
#else
    this->binCountMax = (unsigned int)pow(this->binCount, DIMENSIONS);
#endif
    this->binCount = (unsigned int)pow(this->binCount, DIMENSIONS);

#if defined(MORTON) || defined(HILBERT) ||defined(PEANO)
    printf("Space-filling grid exponent set to: %u\n", this->gridExponent);
#endif
}
void SpatialPartition::setLocationCount(unsigned int t_locationMessageCount)
{
    //Set local copy
    locationMessageCount = t_locationMessageCount;
    //Set device constants
    CUDA_CALL(hipMemcpyToSymbol(HIP_SYMBOL(d_locationMessageCount), &locationMessageCount, sizeof(unsigned int)));
}

void SpatialPartition::launchHashLocationMessages()
{
    int blockSize;   // The launch configurator returned block size 
    CUDA_CALL(hipOccupancyMaxActiveBlocksPerMultiprocessor(&blockSize, hashLocationMessages, 32, 0));//Randomly 32
    // Round up according to array size
    int gridSize = (locationMessageCount + blockSize - 1) / blockSize;
    hashLocationMessages << <gridSize, blockSize >> >(d_keys, d_vals, d_locationMessages);
    CUDA_CALL(hipDeviceSynchronize());
}
int requiredSM_reorderLocationMessages(int blockSize)
{
    return sizeof(unsigned int)*blockSize;
}
#ifdef _DEBUG
void SpatialPartition::launchAssertPBMIntegerity()
{
    int blockSize;   // The launch configurator returned block size 
    CUDA_CALL(hipOccupancyMaxActiveBlocksPerMultiprocessor(&blockSize, hashLocationMessages, 32, 0));//Randomly 32
    // Round up according to array size
    int gridSize = (this->binCountMax + blockSize - 1) / blockSize;
    //Copy messages from d_messages to d_messages_swap, in hash order
    assertPBMIntegrity << <gridSize, blockSize >> >();
    //No sync, called directly after textures have been updated
}
#endif
void SpatialPartition::launchReorderLocationMessages()
{
    int minGridSize, blockSize;   // The launch configurator returned block size 
    hipOccupancyMaxPotentialBlockSizeVariableSMem(&minGridSize, &blockSize, reorderLocationMessages, requiredSM_reorderLocationMessages, 0);
    // Round up according to array size
    int gridSize = (locationMessageCount + blockSize - 1) / blockSize;
    //Copy messages from d_messages to d_messages_swap, in hash order
    reorderLocationMessages << <gridSize, blockSize, requiredSM_reorderLocationMessages(blockSize) >> >(d_keys, d_vals, d_PBM, d_locationMessages, d_locationMessages_swap);
    CUDA_CHECK();
    swap();
    //Wait for return
    CUDA_CALL(hipDeviceSynchronize());
}
void SpatialPartition::swap()
{
    //Switch d_locationMessages and d_locationMessages_swap
    LocationMessages* d_locationmessages_temp = d_locationMessages;
    d_locationMessages = d_locationMessages_swap;
    d_locationMessages_swap = d_locationmessages_temp;
    //Switch hd_locationMessages and hd_locationMessages_swap
    LocationMessages hd_locationmessages_temp = hd_locationMessages;
    hd_locationMessages = hd_locationMessages_swap;
    hd_locationMessages_swap = hd_locationmessages_temp;

#ifdef _DEBUG
    PBM_isBuilt = 0;
    CUDA_CALL(hipMemcpyToSymbol(HIP_SYMBOL(d_PBM_isBuilt), &PBM_isBuilt, sizeof(unsigned int)));
#endif
}
void SpatialPartition::buildPBM()
{
    //If no messages, or instances, don't bother
    if (locationMessageCount<1) return;
#ifdef _DEBUG
    assertSearch();
#endif
    //Fill primitive key/val arrays for sort
    launchHashLocationMessages();
    //Sort key val arrays using thrust/CUB
#ifndef THRUST
    ////CUB version
    //// Determine temporary device storage requirements
    //void *d_temp_storage = NULL;
    //size_t   temp_storage_bytes = 0;
    //hipcub::DeviceRadixSort::SortPairs(d_temp_storage, temp_storage_bytes, d_keys, d_keys_swap, d_vals, d_vals_swap, locationMessageCount);
    //// Allocate temporary storage
    //hipMalloc(&d_temp_storage, temp_storage_bytes);
    // Run sorting operation
    hipcub::DeviceRadixSort::SortPairs(d_CUB_temp_storage, d_CUB_temp_storage_bytes, d_keys, d_keys_swap, d_vals, d_vals_swap, locationMessageCount);
    //Swap arrays
    unsigned int *temp;
    temp = d_keys;
    d_keys = d_keys_swap;
    d_keys_swap = temp;
    temp = d_vals;
    d_vals = d_vals_swap;
    d_vals_swap = temp;
    ////Free temporary memory
    //hipFree(d_temp_storage);
#else
    //Thrust version
    //hipStream_t s1;
    //hipStreamCreate(&s1);
    //thrust::sort_by_key(thrust::cuda::par(s1), d_keys, d_keys + locationMessageCount, d_vals);
    thrust::sort_by_key(thrust::cuda::par, d_keys, d_keys + locationMessageCount, d_vals);
    //hipStreamSynchronize(s1);
    //hipStreamDestroy(s1);
#endif
    CUDA_CALL(hipGetLastError());
    //Reorder map in order of message_hash	
    //Fill pbm start coords with known value 0xffffffff
    //CUDA_CALL(hipMemset(d_PBM, 0xffffffff, PARTITION_GRID_BIN_COUNT * sizeof(int)));
    //Fill pbm end coords with known value 0x00000000 (this should mean if the mysterious bug does occur, the cell is just dropped, not large loop created)
    unsigned int binCount = this->binCountMax;
    CUDA_CALL(hipMemset(d_PBM, 0x00000000, (binCount + 1) * sizeof(unsigned int)));
    launchReorderLocationMessages();
    //Clone data to textures ready for neighbourhood search
    fillTextures();
#ifdef _DEBUG
    launchAssertPBMIntegerity();
    PBM_isBuilt = 1;
    CUDA_CALL(hipMemcpyToSymbol(HIP_SYMBOL(d_PBM_isBuilt), &PBM_isBuilt, sizeof(unsigned int)));
#endif
}
