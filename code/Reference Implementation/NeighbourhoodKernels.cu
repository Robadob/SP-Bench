#include "hip/hip_runtime.h"
#include "NeighbourhoodKernels.cuh"
//getHash already clamps.
//#define SP_NO_CLAMP_GRID //Clamp grid coords to within grid (if it's possible for model to go out of bounds)

__device__ DIMENSIONS_IVEC getGridPosition(DIMENSIONS_VEC worldPos)
{
#ifndef SP_NO_CLAMP_GRID
    //Clamp each grid coord to 0<=x<dim
    return clamp(floor(((worldPos - d_environmentMin) / (d_environmentMax - d_environmentMin))*d_gridDim_float), glm::vec3(0), d_gridDim_float-glm::vec3(1));
#else
    return floor(((worldPos - d_environmentMin) / (d_environmentMax - d_environmentMin))*d_gridDim_float);
#endif
    //#ifdef _3D
    //    glm::ivec3 gridPos;
    //#else
    //    glm::ivec2 gridPos;
    //#endif
    //    gridPos.x = floor(d_gridDim.x * (worldPos.x - d_environmentMin.x) / (d_environmentMax.x - d_environmentMin.x));
    //    gridPos.y = floor(d_gridDim.y * (worldPos.y - d_environmentMin.y) / (d_environmentMax.y - d_environmentMin.y));
    //#ifdef _3D
    //    gridPos.z = floor(d_gridDim.z * (worldPos.z - d_environmentMin.z) / (d_environmentMax.z - d_environmentMin.z));
    //#endif
    //
    //    return gridPos;
}

__device__ int getHash(DIMENSIONS_IVEC gridPos)
{
    //Bound gridPos to gridDimensions
    //Cheaper to bound without mod
    gridPos = clamp(gridPos, DIMENSIONS_IVEC(0), d_gridDim - DIMENSIONS_IVEC(1));
//    gridPos.x = (gridPos.x<0) ? d_gridDim.x - 1 : gridPos.x;
//    gridPos.x = (gridPos.x >= d_gridDim.x) ? 0 : gridPos.x;
//    gridPos.y = (gridPos.y<0) ? d_gridDim.y - 1 : gridPos.y;
//    gridPos.y = (gridPos.y >= d_gridDim.y) ? 0 : gridPos.y;
//#ifdef _3D
//    gridPos.z = (gridPos.z<0) ? d_gridDim.z - 1 : gridPos.z;
//    gridPos.z = (gridPos.z >= d_gridDim.z) ? 0 : gridPos.z;
//#endif

    //Compute hash (effectivley an index for to a bin within the partitioning grid in this case)
    return
#ifdef _3D
        (gridPos.z * d_gridDim.y * d_gridDim.x) +   //z
#endif
        (gridPos.y * d_gridDim.x) +					//y
        gridPos.x; 	                                //x
}
__global__ void hashLocationMessages(unsigned int* keys, unsigned int* vals, LocationMessages* messageBuffer)
{
    int index = (blockIdx.x * blockDim.x) + threadIdx.x;
    //Kill excess threads
    if (index >= d_locationMessageCount) return;

    DIMENSIONS_IVEC gridPos;
    DIMENSIONS_VEC worldPos(
        messageBuffer->locationX[index]
        , messageBuffer->locationY[index]
#ifdef _3D
        , messageBuffer->locationZ[index]
#endif
        );
    gridPos = getGridPosition(worldPos);
    unsigned int hash = getHash(gridPos);
    keys[index] = hash;
    vals[index] = index;
}

__global__ void reorderLocationMessages(
    unsigned int *keys,
    unsigned int *vals,
    unsigned int *pbm,
    LocationMessages *unordered_messages,
    LocationMessages *ordered_messages
    )
{
    extern __shared__ int sm_data[];

    int index = (blockIdx.x * blockDim.x) + threadIdx.x;
    int indexPlus1 = index + 1;

    //Load current key and copy it into shared
    unsigned int key;
    unsigned int old_pos;
    if (index < d_locationMessageCount)
    {//Don't go out of bounds when buffer is at max capacity
        key = keys[index];
        old_pos = vals[index];
        //Every valid thread put key into shared memory
        sm_data[threadIdx.x] = key;
    }
    __syncthreads();
    //Kill excess threads
    if (index >= d_locationMessageCount) return;

    //Load next key
    unsigned int next_key;
    //if thread is final thread
    if (index == d_locationMessageCount-1)
    {
        next_key = d_binCount;
    }
    //If thread is last in block, no next in SM, goto global
    else if (threadIdx.x == blockDim.x-1)
    {
         next_key = keys[indexPlus1];
    }
    else
    {
        next_key = sm_data[threadIdx.x + 1];
    }

    //Set partition boundaries
    //if (index == 0)
    //{//First message, set first bin start
    //    pbm->start[key] = index;
    //}
    //else 
    //if (index == 0)
    //{//First thread, set all bins prior to my key to 0
    //    if (key>0)
    //        for (int k = 0; k < key; k++)
    //            pbm[k] = 0;
    //}
#if _DEBUG
    if (next_key > 125)
        printf("ERROR: PBM generated a next_key that is too high.");
#endif
    if (next_key != key)
    {//Boundary message, set all keys after ours until (inclusive) next_key to our index +1
        for (int k = next_key; k > key; k--)
            pbm[k] = indexPlus1;
    }

    //Order messages into swap space
    ordered_messages->locationX[index] = unordered_messages->locationX[old_pos];
    ordered_messages->locationY[index] = unordered_messages->locationY[old_pos];
#ifdef _3D
    ordered_messages->locationZ[index] = unordered_messages->locationZ[old_pos];
#endif
#ifdef _GL
    ordered_messages->count[index] = unordered_messages->count[old_pos];
#endif
}

__device__ LocationMessage *LocationMessages::getNextNeighbour(LocationMessage *message)
{
    extern __shared__ LocationMessage sm_messages[];
    //LocationMessage *sm_message = &(sm_messages[threadIdx.x]);

    return loadNextMessage();
}
__device__ bool invalidBinYZ(glm::ivec3 bin)
{
    if (
        bin.y<0 || bin.y >= d_gridDim.y ||
        bin.z<0 || bin.z >= d_gridDim.z
        )
    {
        return true;
    }
    return false;
}
__device__ bool invalidBinX(glm::ivec3 bin)
{
    if (
        bin.x<0 || bin.x >= d_gridDim.x 
        )
    {
        return true;
    }
    return false;
}
__device__ bool LocationMessages::nextBin()
{
    extern __shared__ LocationMessage sm_messages[];
    LocationMessage *sm_message = &(sm_messages[threadIdx.x]);

#ifdef _3D
    if (sm_message->state.relative.x >= 1)
    {
        sm_message->state.relative.x = -1;

        if (sm_message->state.relative.y >= 1)
        {
            return false;
        }
        else
        {
            sm_message->state.relative.y++;
        }
    }
    else
    {
        sm_message->state.relative.x++;
    }
    return true;
#else
    if (sm_message->state.relative >= 1)
    {
        return false;
    }
    else
    {
        sm_message->state.relative++;
    }
    return true;
#endif
}
//Load the next desired message into shared memory
__device__ LocationMessage *LocationMessages::loadNextMessage()
{
    extern __shared__ LocationMessage sm_messages[];
    LocationMessage *sm_message = &(sm_messages[threadIdx.x]);

    bool changeBin = true;
    sm_message->state.binIndex++;
    if (sm_message->state.binIndex < sm_message->state.binIndexMax)
        changeBin = false;

    while (changeBin)
    {
        if (nextBin())
        {
            //calculate the next strip of contiguous bins
#ifdef _3D
            glm::ivec3 next_bin_first = sm_message->state.location + glm::ivec3(-1, sm_message->state.relative.x, sm_message->state.relative.y);
#else
            glm::ivec2 next_bin_first = sm_message->state.location + glm::ivec2(-1, sm_message->state.relative);
#endif
            
            DIMENSIONS_IVEC next_bin_last = next_bin_first;
            next_bin_last.x += 2;
            bool firstInvalid = invalidBinX(next_bin_first);
            bool lastInvalid = invalidBinX(next_bin_last);
            if (invalidBinYZ(next_bin_first))
            {//Whole strip invalid, skip
                continue;
            }
            if (firstInvalid)
            {
                next_bin_first.x = 0;
            }
            if (lastInvalid)
            {//If strip ends out of bounds only
                next_bin_last.x = d_gridDim.x-1;//Max x coord
            }

            int next_bin_first_hash = getHash(next_bin_first);
            int next_bin_last_hash = next_bin_first_hash + (next_bin_last.x-next_bin_first.x);//Strips are at most length 3
            if (next_bin_last_hash>getHash(next_bin_last))
            {
                printf("#%i, #%i,(%i +(%i-%i))\n", next_bin_last_hash, getHash(next_bin_last), next_bin_first_hash, next_bin_last.x, next_bin_first.x);
            }

            //use the hash to calculate the start index (pbm stores location of 1st item)
            sm_message->state.binIndex = tex1Dfetch<unsigned int>(d_tex_PBM, next_bin_first_hash);
            sm_message->state.binIndexMax = tex1Dfetch<unsigned int>(d_tex_PBM, next_bin_last_hash+1);
            
            if (sm_message->state.binIndex < sm_message->state.binIndexMax)//(bin_index_min != 0xffffffff)
            {
                break;//Bin strip has items!
            }
        }
        else
        {
            return 0;//All bins exhausted
        }
    }
    sm_message->id = sm_message->state.binIndex;//Duplication of data TODO remove stateBinIndex
    sm_message->location.x = tex1Dfetch<float>(d_tex_location[0], sm_message->state.binIndex);
    sm_message->location.y = tex1Dfetch<float>(d_tex_location[1], sm_message->state.binIndex);
#ifdef _3D
    sm_message->location.z = tex1Dfetch<float>(d_tex_location[2], sm_message->state.binIndex);
#endif

    return sm_message;
}


__device__ LocationMessage *LocationMessages::getFirstNeighbour(DIMENSIONS_VEC location)
{
    extern __shared__ LocationMessage sm_messages[];
    LocationMessage *sm_message = &(sm_messages[threadIdx.x]);
#ifdef _DEBUG
    //If first thread and PBM isn't built, print warning
    if (!d_PBM_isBuilt && (((blockIdx.x * blockDim.x) + threadIdx.x)) == 0)
        printf("PBM has not been rebuilt after calling swap()!\n");
#endif
    sm_message->state.location = getGridPosition(location);
    sm_message->state.binIndex = 0;//Init binIndex greater than equal to binIndexMax to force bin change
    sm_message->state.binIndexMax = 0;
    //Location in moore neighbourhood
    //Start out of range, so we get moved into 1st cell
#ifdef _3D
    sm_message->state.relative = glm::ivec2(-2, -1);
#else
    sm_message->state.relative = -2;
#endif
#ifdef _DEBUG
    LocationMessage *lm = loadNextMessage();
    //if (d_PBM_isBuilt && (((blockIdx.x * blockDim.x) + threadIdx.x)) == 0)
    //{
    //    DIMENSIONS_IVEC pos = getGridPosition(location);
    //    int hash = getHash(pos);
    //    printf("GridDim(%i,%i,%i) Hash(%i) GridPos(%i,%i,%i)\n", d_gridDim.x, d_gridDim.y, d_gridDim.z, hash, pos.x, pos.y, pos.z);
    //}
    if (lm==0)
    {
        //printf("ERROR: getFirstNeighbour() ret 0\n");
    }
    return lm;
#else
    return loadNextMessage();
#endif
}
