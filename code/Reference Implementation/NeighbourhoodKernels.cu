#include "hip/hip_runtime.h"
#include "NeighbourhoodKernels.cuh"
//getHash already clamps.
#define SP_NO_CLAMP_GRID //Clamp grid coords to within grid (if it's possible for model to go out of bounds)

__device__ DIMENSIONS_IVEC getGridPosition(DIMENSIONS_VEC worldPos)
{
#ifndef SP_NO_CLAMP_GRID
    //Clamp each grid coord to 0<=x<dim
    return clamp(floor(((worldPos - d_environmentMin) / (d_environmentMax - d_environmentMin))*d_gridDim_float), glm::vec3(0), d_gridDim_float-glm::vec3(1));
#else
    return floor(((worldPos - d_environmentMin) / (d_environmentMax - d_environmentMin))*d_gridDim_float);
#endif
    //#ifdef _3D
    //    glm::ivec3 gridPos;
    //#else
    //    glm::ivec2 gridPos;
    //#endif
    //    gridPos.x = floor(d_gridDim.x * (worldPos.x - d_environmentMin.x) / (d_environmentMax.x - d_environmentMin.x));
    //    gridPos.y = floor(d_gridDim.y * (worldPos.y - d_environmentMin.y) / (d_environmentMax.y - d_environmentMin.y));
    //#ifdef _3D
    //    gridPos.z = floor(d_gridDim.z * (worldPos.z - d_environmentMin.z) / (d_environmentMax.z - d_environmentMin.z));
    //#endif
    //
    //    return gridPos;
}

__device__ int getHash(DIMENSIONS_IVEC gridPos)
{
    //Bound gridPos to gridDimensions
    //Cheaper to bound without mod
    gridPos = clamp(gridPos, DIMENSIONS_IVEC(0), d_gridDim - DIMENSIONS_IVEC(1));
//    gridPos.x = (gridPos.x<0) ? d_gridDim.x - 1 : gridPos.x;
//    gridPos.x = (gridPos.x >= d_gridDim.x) ? 0 : gridPos.x;
//    gridPos.y = (gridPos.y<0) ? d_gridDim.y - 1 : gridPos.y;
//    gridPos.y = (gridPos.y >= d_gridDim.y) ? 0 : gridPos.y;
//#ifdef _3D
//    gridPos.z = (gridPos.z<0) ? d_gridDim.z - 1 : gridPos.z;
//    gridPos.z = (gridPos.z >= d_gridDim.z) ? 0 : gridPos.z;
//#endif

    //Compute hash (effectivley an index for to a bin within the partitioning grid in this case)
    return
#ifdef _3D
        (gridPos.z * d_gridDim.y * d_gridDim.x) +   //z
#endif
        (gridPos.y * d_gridDim.x) +					//y
        gridPos.x; 	                                //x
}
__global__ void hashLocationMessages(unsigned int* keys, unsigned int* vals, LocationMessages* messageBuffer)
{
    int index = (blockIdx.x * blockDim.x) + threadIdx.x;
    //Kill excess threads
    if (index >= d_locationMessageCount) return;

    DIMENSIONS_IVEC gridPos;
    DIMENSIONS_VEC worldPos(
        messageBuffer->locationX[index]
        , messageBuffer->locationY[index]
#ifdef _3D
        , messageBuffer->locationZ[index]
#endif
        );
    gridPos = getGridPosition(worldPos);
    unsigned int hash = getHash(gridPos);
    keys[index] = hash;
    vals[index] = index;
}

//For-each location message in memory
//Check whether preceding key is the same
__global__ void reorderLocationMessages(
    unsigned int *keys,
    unsigned int *vals,
    unsigned int *pbm,
    LocationMessages *unordered_messages,
    LocationMessages *ordered_messages
    )
{
    extern __shared__ int sm_data[];

    int index = (blockIdx.x * blockDim.x) + threadIdx.x;

    //Load current key and copy it into shared
    unsigned int key;
    unsigned int old_pos;
    if (index < d_locationMessageCount)
    {//Don't go out of bounds when buffer is at max capacity
        key = keys[index];
        old_pos = vals[index];
        //Every valid thread put hash into shared memory
        sm_data[threadIdx.x] = key;
    }
    __syncthreads();
    //Kill excess threads
    if (index >= d_locationMessageCount) return;

    //Load previous key
    unsigned int prev_key = key;//0
    //If thread 0, no prev in warp, goto global
    if (threadIdx.x == 0)
    {
        //Skip if first thread globally
        if (index != 0)
            prev_key = keys[index - 1];
    }
    else
    {
        prev_key = sm_data[threadIdx.x - 1];
    }

    //Set partition boundaries
    //if (index == 0)
    //{//First message, set first bin start
    //    pbm->start[key] = index;
    //}
    //else 
    if (prev_key != key)
    {//Boundary message, update (//start and) ends of boundary
        //    pbm->start[key] = index;
        for (int k = prev_key; k < key;k++)//Loop here stops empty bins being left at 0
            pbm[k] = index;
    }
    if (index == (d_locationMessageCount - 1))
    {//Last message, set last bin end
        pbm[key] = index + 1;
    }

    //Order messages into swap space
    ordered_messages->locationX[index] = unordered_messages->locationX[old_pos];
    ordered_messages->locationY[index] = unordered_messages->locationY[old_pos];
#ifdef _3D
    ordered_messages->locationZ[index] = unordered_messages->locationZ[old_pos];
#endif
}



__device__ LocationMessage *LocationMessages::getNextNeighbour(LocationMessage *message)
{
    extern __shared__ LocationMessage sm_messages[];
    //LocationMessage *sm_message = &(sm_messages[threadIdx.x]);

    return loadNextMessage();
}
__device__ bool LocationMessages::nextBin()
{
    extern __shared__ LocationMessage sm_messages[];
    LocationMessage *sm_message = &(sm_messages[threadIdx.x]);

#ifdef _3D
    if (sm_message->state.relative.x >= 1)
    {
        sm_message->state.relative.x = -1;

        if (sm_message->state.relative.y >= 1)
        {
            return false;
        }
        else
        {
            sm_message->state.relative.y++;
        }
    }
    else
    {
        sm_message->state.relative.x++;
    }
    return true;
#else
    if (sm_message->state.relative >= 1)
    {
        return false;
    }
    else
    {
        sm_message->state.relative++;
    }
    return true;
#endif
}
//Load the next desired message into shared memory
__device__ LocationMessage *LocationMessages::loadNextMessage()
{
    extern __shared__ LocationMessage sm_messages[];
    LocationMessage *sm_message = &(sm_messages[threadIdx.x]);

    bool changeBin = true;
    sm_message->state.binIndex++;
    if (sm_message->state.binIndex < sm_message->state.binIndexMax)
        changeBin = false;

    while (changeBin)
    {
        if (nextBin())
        {
            //calculate the next strip of contiguous bins
#ifdef _3D
            glm::ivec3 next_bin_first = sm_message->state.location + glm::ivec3(-1, sm_message->state.relative.x, sm_message->state.relative.y);
#else
            glm::ivec2 next_bin_first = sm_message->state.location + glm::ivec2(-1, sm_message->state.relative);
#endif
            int next_bin_first_hash = getHash(next_bin_first);
            int next_bin_last_hash = next_bin_first_hash + 2;//Strips are length 3
            //use the hash to calculate the start index (pbm stores location of
            if (next_bin_last_hash >= d_binCount)
                next_bin_last_hash = d_binCount - 1;
            sm_message->state.binIndex = tex1Dfetch<unsigned int>(d_tex_PBM, next_bin_first_hash - 1);
            sm_message->state.binIndexMax = tex1Dfetch<unsigned int>(d_tex_PBM, next_bin_last_hash);
            
            if (sm_message->state.binIndex < sm_message->state.binIndexMax)//(bin_index_min != 0xffffffff)
            {
                break;
            }
            continue;//Strip is empty, continue
        }
        else
        {
            return 0;//All bins exhausted
        }
    }
    sm_message->id = sm_message->state.binIndex;//Duplication of data TODO remove stateBinIndex
    //From texture
    sm_message->location.x = tex1Dfetch<float>(d_tex_location[0], sm_message->state.binIndex);
    sm_message->location.y = tex1Dfetch<float>(d_tex_location[1], sm_message->state.binIndex);
#ifdef _3D
    sm_message->location.z = tex1Dfetch<float>(d_tex_location[2], sm_message->state.binIndex);
#endif

    return sm_message;
}


__device__ LocationMessage *LocationMessages::getFirstNeighbour(DIMENSIONS_VEC location)
{
    extern __shared__ LocationMessage sm_messages[];
    LocationMessage *sm_message = &(sm_messages[threadIdx.x]);

#ifdef _DEBUG
    //If first thread and PBM isn't built, print warning
    if (!d_PBM_isBuilt && (((blockIdx.x * blockDim.x) + threadIdx.x)) == 0)
        printf("PBM has not been rebuilt after calling swap()!\n");
#endif
    sm_message->state.location = getGridPosition(location);
    sm_message->state.binIndex = 0;//Init binIndex greater than equal to binIndexMax to force bin change
    sm_message->state.binIndexMax = 0;
    //Location in moore neighbourhood
    //Start out of range, so we get moved into 1st cell
#ifdef _3D
    sm_message->state.relative = glm::ivec2(-2, -1);
#else
    sm_message->state.relative = -2;
#endif

    return loadNextMessage();
}
