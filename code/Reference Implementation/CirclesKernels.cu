#include "hip/hip_runtime.h"
#include "CirclesKernels.cuh"

__device__ __constant__ float d_attract;
__device__ __constant__ float d_repulse;

__global__ void init_curand(hiprandState *state, unsigned long long seed) {
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    if (id < d_locationMessageCount)
        hiprand_init(seed, id, 0, &state[id]);
}
__global__ void init_particles(hiprandState *state, LocationMessages *locationMessages) {
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    if (id >= d_locationMessageCount)
        return;
    //curand_unform returns 0<x<=1.0, not much can really do about 0 exclusive
    //negate and  + 1.0, to make  0<=x<1.0
    locationMessages->locationX[id] = (-hiprand_uniform(&state[id]) + 1.0f)*d_environmentMax.x;
    locationMessages->locationY[id] = (-hiprand_uniform(&state[id]) + 1.0f)*d_environmentMax.y;
#ifdef _3D
    locationMessages->locationZ[id] = (-hiprand_uniform(&state[id]) + 1.0f)*d_environmentMax.z;
#endif
}

__global__ void step_model(LocationMessages *locationMessagesIn, LocationMessages *locationMessagesOut)
{

    int id = blockIdx.x * blockDim.x + threadIdx.x;
    if (id >= d_locationMessageCount)
        return;

    //Get my local location
#ifdef _3D
    glm::vec3 myLoc(locationMessagesIn->locationX[id], locationMessagesIn->locationY[id], locationMessagesIn->locationZ[id]), theirLoc, locDiff;
#else
    glm::vec2 myLoc(locationMessagesIn->locationX[id], locationMessagesIn->locationY[id]), theirLoc, locDiff;
#endif
    //Get first message
    float dist, separation, k;
    LocationMessage *lm = locationMessagesIn->getFirstNeighbour(myLoc);
    //Always atleast 1 location message, our own location!
    int counter = 0;
    do
    {
        counter++;
        if ((lm->id != id))
        {
            locDiff = myLoc - lm->location;//Difference
            if (locDiff==DIMENSIONS_VEC(0))//Ignore distance 0
            {
                lm = locationMessagesIn->getNextNeighbour(lm);
                continue;
            }
            theirLoc = locDiff*locDiff;//Squared
            dist = sqrt(glm::compAdd(theirLoc));//Distance (via pythagoras)
            separation = dist - d_interactionRad - d_interactionRad;
            if (separation < d_interactionRad)
            {
                if (separation > 0.0f)
                    k = d_attract;
                else
                    k = d_repulse;
                myLoc += (k*separation*(locDiff / dist));
            }
        }
        lm = locationMessagesIn->getNextNeighbour(lm);//Returns a pointer to shared memory or 0
    } while (lm);
    //Export myloc?
    locationMessagesOut->locationX[id] = myLoc.x;
    locationMessagesOut->locationY[id] = myLoc.y;
#ifdef _3D
    locationMessagesOut->locationZ[id] = myLoc.z;
#endif
#if defined(_GL) || defined(_DEBUG)
    locationMessagesOut->count[id] = counter/(float)d_locationMessageCount;
  //  printf("%.3f\n", locationMessagesOut->count[id]);
#endif
}
