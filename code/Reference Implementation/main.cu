#include "hip/hip_runtime.h"
#include "Neighbourhood.cuh"
#include "Circles.cuh"
#include "Visualisation/Visualisation.h"
#include "ParticleScene.h"
//#include <hip/hip_runtime.h>
//#include <>

__global__ void initLocations(
    LocationMessages *messages
    )
{

    int index = __mul24(blockIdx.x, blockDim.x) + threadIdx.x;

    if (index >= d_locationMessageCount) return;

    //messages->locationX[index] = ;
   // messages->locationY[index] = ;
#ifdef _3D
   // messages->locationZ[index] = ;
#endif
}


int main()
{
    hipSetDevice(0);
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);

    const unsigned int width = 50;
    const float density = 0.005f;
    const float interactionRad = 10.0f;
    const float attractionForce = 0.0001f;
    const float repulsionForce = 0.0001f;
    const unsigned long long iterations = 10000;

    Visualisation v("Visulisation Example", 1280, 720);
    Circles<SpatialPartition> model(width, density, interactionRad, attractionForce, repulsionForce);
    const Time_Init initTimes = model.initPopulation();//Need to init textures before creating the scene
    ParticleScene<SpatialPartition> *scene = new ParticleScene<SpatialPartition>(v, model);

    //Init model
    printf("Init Complete - Times\n");
    printf("CuRand init - %.3fs\n", initTimes.initCurand / 1000);
    printf("Main kernel - %.3fs\n", initTimes.kernel / 1000);
    printf("Build PBM   - %.3fs\n", initTimes.pbm / 1000);
    printf("CuRand free - %.3fs\n", initTimes.freeCurand / 1000);
    printf("Combined    - %.3fs\n", initTimes.overall / 1000);
    printf("\n");
    //Start visualisation
    //v.runAsync();
    //v.run();
    //Do iterations
    Time_Step_dbl average = {};//init

    printf("\n");
    for (unsigned long long i = 0; i < iterations; i++)
    {
        const Time_Step iterTime = model.step();
        //Pass count to visualisation
        scene->setCount(model.getPartition()->getLocationCount());
        //Calculate averages
        average.overall += iterTime.overall / iterations;
        average.kernel += iterTime.kernel / iterations;
        average.texture += iterTime.texture / iterations;
        v.render();
        printf("\r%6llu/%llu", i, iterations);
    }
    printf("Model complete - Average Times\n");
    printf("Main kernel - %.3fs\n", average.kernel / 1000);
    printf("Build PBM   - %.3fs\n", average.texture / 1000);
    printf("Combined    - %.3fs\n", average.overall / 1000);
    printf("\n");

    //Calculate final timing
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    float totalTime;
    hipEventElapsedTime(&totalTime, start, stop);

    printf("Total Runtime: %.3fs\n", totalTime * 1000);

    v.run();

    //Wait for input before exit
    getchar();
    return 0;
}
