#include "hip/hip_runtime.h"
#include "Neighbourhood.cuh"
#include "NeighbourhoodConstants.cuh"
#include "NeighbourhoodKernels.cuh"
#ifndef THRUST
#include <hipcub/hipcub.hpp>
#else
#include <thrust/sort.h>
#include <thrust/system/cuda/execution_policy.h>
#endif
#ifdef _GL
#include <cuda_gl_interop.h>
#endif
#ifdef _DEBUG
#include <glm/gtc/epsilon.hpp>
#endif
SpatialPartition::SpatialPartition(DIMENSIONS_VEC  environmentMin, DIMENSIONS_VEC environmentMax, unsigned int maxAgents, float interactionRad)
    : environmentMin(environmentMin)
    , environmentMax(environmentMax)
    , maxAgents(maxAgents)
    , interactionRad(interactionRad)
    , locationMessageCount(0)
    , gridDim((environmentMax - environmentMin) / interactionRad)
#ifdef _DEBUG
    , PBM_isBuilt(0)
#endif
{
    //Allocate bins in GPU memory
    deviceAllocateLocationMessages(&d_locationMessages);
    //Allocate bins swap in GPU memory
    deviceAllocateLocationMessages(&d_locationMessages_swap);
    //Allocate PBM
    deviceAllocatePBM(&d_PBM);
    //Allocate primitive structures
    deviceAllocatePrimitives(&d_keys, &d_vals);
#ifndef THRUST
    deviceAllocatePrimitives(&d_keys_swap, &d_vals_swap);
#endif
    //Allocate tex
    deviceAllocateTextures();
    //Set device constants
    CUDA_CALL(hipMemcpyToSymbol(HIP_SYMBOL(d_interactionRad), &interactionRad, sizeof(float)));
    CUDA_CALL(hipMemcpyToSymbol(HIP_SYMBOL(d_gridDim), &gridDim, sizeof(DIMENSIONS_IVEC)));
    DIMENSIONS_VEC t_gridDim = (DIMENSIONS_VEC)gridDim;
    CUDA_CALL(hipMemcpyToSymbol(HIP_SYMBOL(d_gridDim_float), &t_gridDim, sizeof(DIMENSIONS_VEC)));

    CUDA_CALL(hipMemcpyToSymbol(HIP_SYMBOL(d_environmentMin), &environmentMin, sizeof(DIMENSIONS_VEC)));
    CUDA_CALL(hipMemcpyToSymbol(HIP_SYMBOL(d_environmentMax), &environmentMax, sizeof(DIMENSIONS_VEC)));

#ifdef _DEBUG
    CUDA_CALL(hipMemcpyToSymbol(HIP_SYMBOL(d_PBM_isBuilt), &PBM_isBuilt, sizeof(unsigned int)));
#endif
    setLocationCount(locationMessageCount);
    unsigned int t_binCount = getBinCount();
    CUDA_CALL(hipMemcpyToSymbol(HIP_SYMBOL(d_binCount), &t_binCount, sizeof(unsigned int)));
}
SpatialPartition::~SpatialPartition()
{
    //Dellocate bins in GPU memory
    deviceDeallocateLocationMessages(d_locationMessages);
    //Dellocate bins swap in GPU memory
    deviceDeallocateLocationMessages(d_locationMessages_swap);
    //Dellocate PBM
    deviceDeallocatePBM(d_PBM);
    //Deallocated primitive structures
    deviceDeallocatePrimitives(d_keys, d_vals);
#ifndef THRUST
    deviceDeallocatePrimitives(d_keys_swap, d_vals_swap);
#endif
    //Deallocate tex
    deviceDeallocateTextures();
}
#ifdef _DEBUG

DIMENSIONS_IVEC SpatialPartition::getGridPosition(DIMENSIONS_VEC worldPos)
{
#ifndef SP_NO_CLAMP_GRID
    //Clamp each grid coord to 0<=x<dim
    return clamp(floor(((worldPos - environmentMin) / (environmentMax - environmentMin))*glm::vec3(gridDim)), glm::vec3(0), glm::vec3(gridDim)-glm::vec3(1));
#else
    return floor(((worldPos - environmentMin) / (environmentMax - environmentMin))*glm::vec3(gridDim));
#endif
}

int SpatialPartition::getHash(DIMENSIONS_IVEC gridPos)
{
    gridPos = clamp(gridPos, DIMENSIONS_IVEC(0), gridDim - DIMENSIONS_IVEC(1));
    return
#ifdef _3D
        (gridPos.z * gridDim.y * gridDim.x) +   //z
#endif
        (gridPos.y * gridDim.x) +					//y
        gridPos.x; 	                                //x
}
DIMENSIONS_IVEC SpatialPartition::getPos(unsigned int hash)
{
    if (hash >= getBinCount())
        return DIMENSIONS_IVEC(-1);
    else
    {
#ifdef _3D

        int z = (hash / (gridDim.y * gridDim.x));
        int y = (hash % (gridDim.y * gridDim.x)) / gridDim.x;
        int x = (hash % (gridDim.y * gridDim.x)) % gridDim.x;
        return DIMENSIONS_IVEC(x, y, z);
#else
        int y = hash / gridDim.x;
        int x = hash % gridDim.x;
        return DIMENSIONS_IVEC(x, y);
#endif
    }
}
bool SpatialPartition::isValid(DIMENSIONS_IVEC bin) const
{
    if (
#ifdef _3D
        bin.z<0 || bin.z >= gridDim.z ||
#endif
        bin.y<0 || bin.y >= gridDim.y ||
        bin.x<0 || bin.x >= gridDim.x 
        )
    {
        return false;
    }
    return true;
}
void SpatialPartition::assertSearch()
{
    unsigned int outCount = getBinCount() + 1;
    unsigned int tableSize = ((outCount / 10) + 1) * 10;

    //Copy raw PBM from device to host
    unsigned int *PBM_raw = static_cast<unsigned int *>(malloc(sizeof(unsigned int)*tableSize));
    memset(PBM_raw, 0, tableSize * sizeof(unsigned int));
    CUDA_CALL(hipMemcpy(PBM_raw, d_PBM, sizeof(unsigned int)*outCount, hipMemcpyDeviceToHost));

    //Calculate the size of every bin
    unsigned int *PBM_binSize = static_cast<unsigned int *>(malloc(sizeof(unsigned int)*tableSize));
    for (unsigned int i = 0; i < tableSize; i++)
    {
        if (i < outCount - 1)
            PBM_binSize[i] = PBM_raw[i + 1] - PBM_raw[i];
        else
        {
            PBM_binSize[i] = 11111;
        }

    }

    //Calculate the size of each bin's neighbourhood
    unsigned int *PBM_neighbourhoodSize = static_cast<unsigned int *>(malloc(sizeof(unsigned int)*tableSize));
    for (unsigned int i = 0; i < ((outCount / 10) + 1) * 10; i++)
    {
        PBM_neighbourhoodSize[i] = 0;
        if (i < outCount - 1)
        {
            DIMENSIONS_IVEC curCell = getPos(i);
            for (int x = -1; x <= 1; x++)
                for (int y = -1; y <= 1; y++)
                    for (int z = -1; z <= 1; z++)
                    {
                DIMENSIONS_IVEC neighbourCell = curCell + DIMENSIONS_IVEC(x, y, z);
                if (isValid(neighbourCell))
                {
                    PBM_neighbourhoodSize[i] += PBM_binSize[getHash(neighbourCell)];
                }
                    }
        }

    }

    //Copy every location and neighbour count from device to host
    float *d_bufferPtr;
    LocationMessages lm;
    lm.locationX = (float*)malloc(sizeof(float)*locationMessageCount);
    CUDA_CALL(hipMemcpy(&d_bufferPtr, &d_locationMessages_swap->locationX, sizeof(float*), hipMemcpyDeviceToHost));
    CUDA_CALL(hipMemcpy(lm.locationX, d_bufferPtr, sizeof(float)*locationMessageCount, hipMemcpyDeviceToHost));
    lm.locationY = (float*)malloc(sizeof(float)*locationMessageCount);
    CUDA_CALL(hipMemcpy(&d_bufferPtr, &d_locationMessages_swap->locationY, sizeof(float*), hipMemcpyDeviceToHost));
    CUDA_CALL(hipMemcpy(lm.locationY, d_bufferPtr, sizeof(float)*locationMessageCount, hipMemcpyDeviceToHost));
#ifdef _3D
    lm.locationZ = (float*)malloc(sizeof(float)*locationMessageCount);
    CUDA_CALL(hipMemcpy(&d_bufferPtr, &d_locationMessages_swap->locationZ, sizeof(float*), hipMemcpyDeviceToHost));
    CUDA_CALL(hipMemcpy(lm.locationZ, d_bufferPtr, sizeof(float)*locationMessageCount, hipMemcpyDeviceToHost));
#endif
    lm.count = (float*)malloc(sizeof(float)*locationMessageCount);
    CUDA_CALL(hipMemcpy(&d_bufferPtr, &d_locationMessages_swap->count, sizeof(float*), hipMemcpyDeviceToHost));
    CUDA_CALL(hipMemcpy(lm.count, d_bufferPtr, sizeof(float)*locationMessageCount, hipMemcpyDeviceToHost));
    //ASSERT: Every agent searched the right amount of neighbours
    unsigned int matchFails = 0;
    for (unsigned int i = 0; i < locationMessageCount; i++)
    {
        //For rendering purposes the count is stored as count/totalMessages, invert this math for assertion
        unsigned int hash = getHash(getGridPosition(glm::vec3(lm.locationX[i], lm.locationY[i], lm.locationZ[i])));
        if (glm::epsilonNotEqual(lm.count[i], PBM_neighbourhoodSize[hash] / (float)locationMessageCount, 0.5f))
        {
            //printf("%u=%u-%f=%f,", (unsigned int)(lm.count[i] * locationMessageCount), PBM_neighbourhoodSize[hash], lm.count[i], PBM_neighbourhoodSize[hash] / (float)locationMessageCount);
            matchFails++;
        }
    }
    //Free location/count data
    free(lm.locationX);
    free(lm.locationY);
    free(lm.locationZ);
    free(lm.count);
    if (matchFails>0)
    {
        printf("ERROR: Neighbour search totals do not match (%u/%u)\n", matchFails, locationMessageCount);
    }
    else
    {
        free(PBM_raw);
        free(PBM_binSize);
        free(PBM_neighbourhoodSize);
        return;
    }
    //Output the 3 PBM_ data structures to file in a readable format
    FILE *file = fopen("../logs/PBM.txt", "w");
    fprintf(file, "ERROR: Neighbour search totals do not match (%u/%u)\n", matchFails, locationMessageCount);
    fprintf(file, "Raw PBM\n");
    fprintf(file, "|%5s�%5s|%5s|%5s|%5s|%5s|%5s|%5s|%5s|%5s|%5s|\n", "", "0", "1", "2", "3", "4", "5", "6", "7", "8", "9");
    fprintf(file, "|-----�-----|-----|-----|-----|-----|-----|-----|-----|-----|-----|\n");
    for (unsigned int i = 0; i < (outCount / 10)-1; i++)
    {
        fprintf(file, "|%4u0�%5u|%5u|%5u|%5u|%5u|%5u|%5u|%5u|%5u|%5u|\n", i, 
            PBM_raw[((outCount / 10)*i) + 0],
            PBM_raw[((outCount / 10)*i) + 1],
            PBM_raw[((outCount / 10)*i) + 2],
            PBM_raw[((outCount / 10)*i) + 3],
            PBM_raw[((outCount / 10)*i) + 4],
            PBM_raw[((outCount / 10)*i) + 5],
            PBM_raw[((outCount / 10)*i) + 6],
            PBM_raw[((outCount / 10)*i) + 7],
            PBM_raw[((outCount / 10)*i) + 8],
            PBM_raw[((outCount / 10)*i) + 9]
            );
    }
    fprintf(file, "Bin Size\n");
    fprintf(file, "|%5s�%5s|%5s|%5s|%5s|%5s|%5s|%5s|%5s|%5s|%5s|\n", "", "0", "1", "2", "3", "4", "5", "6", "7", "8", "9");
    fprintf(file, "|-----�-----|-----|-----|-----|-----|-----|-----|-----|-----|-----|\n");
    for (unsigned int i = 0; i < (outCount / 10) - 1; i++)
    {
        fprintf(file, "|%4u0�%5u|%5u|%5u|%5u|%5u|%5u|%5u|%5u|%5u|%5u|\n", i,
            PBM_binSize[((outCount / 10)*i) + 0],
            PBM_binSize[((outCount / 10)*i) + 1],
            PBM_binSize[((outCount / 10)*i) + 2],
            PBM_binSize[((outCount / 10)*i) + 3],
            PBM_binSize[((outCount / 10)*i) + 4],
            PBM_binSize[((outCount / 10)*i) + 5],
            PBM_binSize[((outCount / 10)*i) + 6],
            PBM_binSize[((outCount / 10)*i) + 7],
            PBM_binSize[((outCount / 10)*i) + 8],
            PBM_binSize[((outCount / 10)*i) + 9]
            );
    }
    fprintf(file, "Neighbour Count\n");
    fprintf(file, "|%5s�%5s|%5s|%5s|%5s|%5s|%5s|%5s|%5s|%5s|%5s|\n", "", "0", "1", "2", "3", "4", "5", "6", "7", "8", "9");
    fprintf(file, "|-----�-----|-----|-----|-----|-----|-----|-----|-----|-----|-----|\n");
    for (unsigned int i = 0; i < (outCount / 10) - 1; i++)
    {
        fprintf(file, "|%4u0�%5u|%5u|%5u|%5u|%5u|%5u|%5u|%5u|%5u|%5u|\n", i,
            PBM_neighbourhoodSize[((outCount / 10)*i) + 0],
            PBM_neighbourhoodSize[((outCount / 10)*i) + 1],
            PBM_neighbourhoodSize[((outCount / 10)*i) + 2],
            PBM_neighbourhoodSize[((outCount / 10)*i) + 3],
            PBM_neighbourhoodSize[((outCount / 10)*i) + 4],
            PBM_neighbourhoodSize[((outCount / 10)*i) + 5],
            PBM_neighbourhoodSize[((outCount / 10)*i) + 6],
            PBM_neighbourhoodSize[((outCount / 10)*i) + 7],
            PBM_neighbourhoodSize[((outCount / 10)*i) + 8],
            PBM_neighbourhoodSize[((outCount / 10)*i) + 9]
            );
    }
    //Cleanup resources
    fclose(file);
    free(PBM_raw);
    free(PBM_binSize);
    free(PBM_neighbourhoodSize);
}
#endif
void SpatialPartition::deviceAllocateLocationMessages(LocationMessages **d_locMessage)
{
    unsigned int binCount = getBinCount();
    CUDA_CALL(hipMalloc(d_locMessage, sizeof(LocationMessages)));
    float *d_loc_temp;
    CUDA_CALL(hipMalloc(&d_loc_temp, sizeof(float)*maxAgents));
    CUDA_CALL(hipMemcpy(&((*d_locMessage)->locationX), &d_loc_temp, sizeof(float*), hipMemcpyHostToDevice));
    CUDA_CALL(hipMalloc(&d_loc_temp, sizeof(float)*maxAgents));
    CUDA_CALL(hipMemcpy(&((*d_locMessage)->locationY), &d_loc_temp, sizeof(float*), hipMemcpyHostToDevice));
#ifdef _3D
    CUDA_CALL(hipMalloc(&d_loc_temp, sizeof(float)*maxAgents));
    CUDA_CALL(hipMemcpy(&((*d_locMessage)->locationZ), &d_loc_temp, sizeof(float*), hipMemcpyHostToDevice));
#endif
#if defined(_GL) || defined(_DEBUG)
    CUDA_CALL(hipMalloc(&d_loc_temp, sizeof(float)*maxAgents));
    CUDA_CALL(hipMemcpy(&((*d_locMessage)->count), &d_loc_temp, sizeof(float*), hipMemcpyHostToDevice));
#endif
}
void SpatialPartition::deviceAllocatePBM(unsigned int **d_PBM_t)
{
    unsigned int binCount = getBinCount();
    CUDA_CALL(hipMalloc(d_PBM_t, sizeof(unsigned int)*(binCount+1)));
}
void SpatialPartition::deviceAllocatePrimitives(unsigned int **d_keys, unsigned int **d_vals)
{
    unsigned int binCount = getBinCount();
    CUDA_CALL(hipMalloc(d_keys, sizeof(unsigned int)*maxAgents));
    CUDA_CALL(hipMalloc(d_vals, sizeof(unsigned int)*maxAgents));
}
void SpatialPartition::deviceAllocateTextures()
{
    //Locations
#ifdef _GL
#pragma unroll 3
    for (unsigned int i = 0; i < DIMENSIONS;i++)
        deviceAllocateGLTexture_float(i);
    deviceAllocateGLTexture_float2();//Allocate a texture to store counting info in (Used to colour the visualisation
#else
#pragma unroll 3
    for (unsigned int i = 0; i < DIMENSIONS;i++)
        deviceAllocateTexture_float(i);
#endif
    //PBM
    deviceAllocateTexture_int();
}
void SpatialPartition::fillTextures()
{
    float *d_bufferPtr;
    //Potentially refactor so we store/swap these pointers on host in syncrhonisation
    CUDA_CALL(hipMemcpy(&d_bufferPtr, &d_locationMessages->locationX, sizeof(float*), hipMemcpyDeviceToHost));
    CUDA_CALL(hipMemcpy(tex_loc_ptr[0], d_bufferPtr, locationMessageCount*sizeof(float), hipMemcpyDeviceToDevice));
    CUDA_CALL(hipMemcpy(&d_bufferPtr, &d_locationMessages->locationY, sizeof(float*), hipMemcpyDeviceToHost));
    CUDA_CALL(hipMemcpy(tex_loc_ptr[1], d_bufferPtr, locationMessageCount*sizeof(float), hipMemcpyDeviceToDevice));
#ifdef _3D
    CUDA_CALL(hipMemcpy(&d_bufferPtr, &d_locationMessages->locationZ, sizeof(float*), hipMemcpyDeviceToHost));
    CUDA_CALL(hipMemcpy(tex_loc_ptr[2], d_bufferPtr, locationMessageCount*sizeof(float), hipMemcpyDeviceToDevice));
#endif
#ifdef _GL
    CUDA_CALL(hipMemcpy(&d_bufferPtr, &d_locationMessages->count, sizeof(float*), hipMemcpyDeviceToHost));
    CUDA_CALL(hipMemcpy(tex_location_ptr_count, d_bufferPtr, locationMessageCount*sizeof(float), hipMemcpyDeviceToDevice));
#endif

    CUDA_CALL(hipMemcpy(tex_PBM_ptr, d_PBM, (getBinCount()+1)*sizeof(unsigned int), hipMemcpyDeviceToDevice));
}

void SpatialPartition::deviceAllocateTexture_float(unsigned int i)
{
    if (i >= DIMENSIONS)
        return;
    //Allocate cuda array
    CUDA_CALL(hipMalloc(&tex_loc_ptr[i], maxAgents*sizeof(float)));
    //Define cuda resource from array
    hipResourceDesc resDesc;
    memset(&resDesc, 0, sizeof(hipResourceDesc));
    resDesc.resType = hipResourceTypeLinear;
    resDesc.res.linear.devPtr = tex_loc_ptr[i];
    resDesc.res.linear.desc.f = hipChannelFormatKindFloat;
    resDesc.res.linear.desc.x = 32; // bits per channel
    resDesc.res.linear.sizeInBytes = maxAgents*sizeof(float);
    //Define a cuda texture format
    hipTextureDesc texDesc;
    memset(&texDesc, 0, sizeof(hipTextureDesc));
    texDesc.readMode = hipReadModeElementType;
    //Create texture obj
    CUDA_CALL(hipCreateTextureObject(&tex_location[i], &resDesc, &texDesc, NULL));
    //Copy obj to const memory
    CUDA_CALL(hipMemcpyToSymbol(HIP_SYMBOL(d_tex_location), &tex_location[i], sizeof(hipTextureObject_t), i*sizeof(hipTextureObject_t)));
}
#ifdef _GL
void SpatialPartition::deviceAllocateGLTexture_float(unsigned int i)//GLuint *glTex, GLuint *glTbo, hipGraphicsResource_t *cuGres, hipArray_t *cuArr, hipTextureObject_t *tex, hipTextureObject_t *d_const, const unsigned int size)
{
    if (i >= DIMENSIONS)
        return;
    float *data = new float[maxAgents];
    //Gen tex
    GL_CALL(glGenTextures(1, &gl_tex[i]));
    //Gen buffer
    GL_CALL(glGenBuffers(1, &gl_tbo[i]));
    //Size buffer and tie to tex
    GL_CALL(glBindBuffer(GL_TEXTURE_BUFFER, gl_tbo[i]));
    GL_CALL(glBindBuffer(GL_TEXTURE_BUFFER, gl_tbo[i]));
    GL_CALL(glBufferData(GL_TEXTURE_BUFFER, maxAgents*sizeof(float), 0, GL_STATIC_DRAW));
   
    GL_CALL(glBindTexture(GL_TEXTURE_BUFFER, gl_tex[i]));
    //glBindTexture(GL_TEXTURE_2D, 0);
    GL_CALL(glTexBuffer(GL_TEXTURE_BUFFER, GL_R32F, gl_tbo[i]));
    GL_CALL(glBindBuffer(GL_TEXTURE_BUFFER, 0));
    GL_CALL(glBindTexture(GL_TEXTURE_BUFFER, 0));

    //Get CUDA handle to texture
    memset(&gl_gRes[i], 0, sizeof(hipGraphicsResource_t));
    CUDA_CALL(hipGraphicsGLRegisterBuffer(&gl_gRes[i], gl_tbo[i], cudaGraphicsMapFlagsNone));//GL_TEXTURE_BUFFER IS UNDOCUMENTED
    //Map/convert this to something hipArray_t
    CUDA_CALL(hipGraphicsMapResources(1, &gl_gRes[i]));
    CUDA_CALL(hipGraphicsResourceGetMappedPointer((void**)&tex_loc_ptr[i], 0, gl_gRes[i]));
    CUDA_CALL(hipGraphicsUnmapResources(1, &gl_gRes[i], 0));
    //Create a texture object from the hipArray_t
    hipResourceDesc resDesc;
    memset(&resDesc, 0, sizeof(hipResourceDesc));
    resDesc.resType = hipResourceTypeLinear;
    resDesc.res.linear.devPtr = tex_loc_ptr[i];
    resDesc.res.linear.desc.f = hipChannelFormatKindFloat;
    resDesc.res.linear.desc.x = 32; // bits per channel
    resDesc.res.linear.sizeInBytes = maxAgents*sizeof(float);
    hipTextureDesc texDesc;
    memset(&texDesc, 0, sizeof(hipTextureDesc));
    texDesc.readMode = hipReadModeElementType;
    CUDA_CALL(hipCreateTextureObject(&tex_location[i], &resDesc, &texDesc, nullptr));
    //Copy texture object to device constant
    CUDA_CALL(hipMemcpyToSymbol(HIP_SYMBOL(d_tex_location), &tex_location[i], sizeof(hipTextureObject_t), i*sizeof(hipTextureObject_t)));
    delete data;
}
#endif
/*
Allocates the PBM texture, which is only accessed via CUDA & memcpy
*/
void SpatialPartition::deviceAllocateTexture_int()
{
    //Define cuda array format
    //Allocate cuda array
    unsigned int size = getBinCount() + 1;
    CUDA_CALL(hipMalloc(&tex_PBM_ptr, size*sizeof(unsigned int)));//Number of elements, not bytes
    //Define cuda resource from array
    hipResourceDesc resDesc;
    memset(&resDesc, 0, sizeof(hipResourceDesc));
    resDesc.resType = hipResourceTypeLinear;
    resDesc.res.linear.devPtr = tex_PBM_ptr;
    resDesc.res.linear.desc.f = hipChannelFormatKindUnsigned;
    resDesc.res.linear.desc.x = 32; // bits per channel
    resDesc.res.linear.sizeInBytes = size*sizeof(unsigned int);

    hipTextureDesc texDesc;
    memset(&texDesc, 0, sizeof(hipTextureDesc));
    texDesc.readMode = hipReadModeElementType;

    CUDA_CALL(hipCreateTextureObject(&tex_PBM, &resDesc, &texDesc, NULL));
    CUDA_CALL(hipMemcpyToSymbol(HIP_SYMBOL(d_tex_PBM), &tex_PBM, sizeof(hipTextureObject_t)));
}
#ifdef _GL
/*
Allocates the count texture, which is only accessed via memcpy & GL
*/
void SpatialPartition::deviceAllocateGLTexture_float2()
{
    int *data = new int[maxAgents];
    //Gen tex
    GL_CALL(glGenTextures(1, &gl_tex_count));
    //Gen buffer
    GL_CALL(glGenBuffers(1, &gl_tbo_count));
    //Size buffer and tie to tex
    GL_CALL(glBindBuffer(GL_TEXTURE_BUFFER, gl_tbo_count));
    GL_CALL(glBindBuffer(GL_TEXTURE_BUFFER, gl_tbo_count));
    GL_CALL(glBufferData(GL_TEXTURE_BUFFER, maxAgents*sizeof(float), 0, GL_STATIC_DRAW));

    GL_CALL(glBindTexture(GL_TEXTURE_BUFFER, gl_tex_count));
    //glBindTexture(GL_TEXTURE_2D, 0);
    GL_CALL(glTexBuffer(GL_TEXTURE_BUFFER, GL_R32F, gl_tbo_count));
    GL_CALL(glBindBuffer(GL_TEXTURE_BUFFER, 0));
    GL_CALL(glBindTexture(GL_TEXTURE_BUFFER, 0));

    //Get CUDA handle to texture
    memset(&gl_gRes_count, 0, sizeof(hipGraphicsResource_t));
    CUDA_CALL(hipGraphicsGLRegisterBuffer(&gl_gRes_count, gl_tbo_count, cudaGraphicsMapFlagsNone));
    //Map/convert this to something hipArray_t
    CUDA_CALL(hipGraphicsMapResources(1, &gl_gRes_count));
    CUDA_CALL(hipGraphicsResourceGetMappedPointer((void**)&tex_location_ptr_count, 0, gl_gRes_count));
    CUDA_CALL(hipGraphicsUnmapResources(1, &gl_gRes_count, 0));
    delete data;
}
#endif
void SpatialPartition::deviceDeallocateLocationMessages(LocationMessages *d_locMessage)
{
    float *d_loc_temp;
    CUDA_CALL(hipMemcpy(&d_loc_temp, d_locMessage->locationX, sizeof(float*), hipMemcpyHostToDevice));
    CUDA_CALL(hipFree(d_loc_temp));
    CUDA_CALL(hipMemcpy(&d_loc_temp, d_locMessage->locationY, sizeof(float*), hipMemcpyHostToDevice));
    CUDA_CALL(hipFree(d_loc_temp));
#ifdef _3D
    CUDA_CALL(hipMemcpy(d_loc_temp, d_locMessage->locationZ, sizeof(float*), hipMemcpyHostToDevice));
    CUDA_CALL(hipFree(d_loc_temp));
#endif
    CUDA_CALL(hipFree(d_locMessage));
}
void SpatialPartition::deviceDeallocatePBM(unsigned int *d_PBM_t)
{
    CUDA_CALL(hipFree(d_PBM_t));
}
void SpatialPartition::deviceDeallocatePrimitives(unsigned int *d_keys, unsigned int *d_vals)
{
    CUDA_CALL(hipFree(d_keys));
    CUDA_CALL(hipFree(d_vals));
}
void SpatialPartition::deviceDeallocateTextures()
{

#pragma unroll
    for (unsigned int i = 0; i < DIMENSIONS; i++)
    {
        hipDestroyTextureObject(tex_location[i]);
#ifdef _GL
        hipGraphicsUnregisterResource(gl_gRes[i]);
        GL_CALL(glDeleteBuffers(1, &gl_tbo[i]));
        GL_CALL(glDeleteTextures(1, &gl_tex[i]));
#else
        hipFree(tex_loc_ptr[i]);
#endif
    }
    hipDestroyTextureObject(tex_PBM);
    hipFree(tex_PBM_ptr);
#ifdef _GL
    hipGraphicsUnregisterResource(gl_gRes_count);
    GL_CALL(glDeleteBuffers(1, &gl_tbo_count));
    GL_CALL(glDeleteTextures(1, &gl_tex_count));
#endif
}

unsigned int SpatialPartition::getBinCount()
{
    return (unsigned int)glm::compMul(gridDim);
}
void SpatialPartition::setLocationCount(unsigned int t_locationMessageCount)
{
    //Set local copy
    locationMessageCount = t_locationMessageCount;
    //Set device constants
    CUDA_CALL(hipMemcpyToSymbol(HIP_SYMBOL(d_locationMessageCount), &locationMessageCount, sizeof(unsigned int)));
}

void SpatialPartition::launchHashLocationMessages()
{
    int blockSize;   // The launch configurator returned block size 
    CUDA_CALL(hipOccupancyMaxActiveBlocksPerMultiprocessor(&blockSize, hashLocationMessages, 32, 0));//Randomly 32
    // Round up according to array size
    int gridSize = (locationMessageCount + blockSize - 1) / blockSize;
    hashLocationMessages <<<gridSize, blockSize>>>(d_keys, d_vals, d_locationMessages);
    CUDA_CALL(hipDeviceSynchronize());
}
int requiredSM_reorderLocationMessages(int blockSize)
{
    return sizeof(unsigned int)*blockSize;
}
void SpatialPartition::launchReorderLocationMessages()
{
    int minGridSize, blockSize;   // The launch configurator returned block size 
    hipOccupancyMaxPotentialBlockSizeVariableSMem(&minGridSize, &blockSize, reorderLocationMessages, requiredSM_reorderLocationMessages, 0);
    // Round up according to array size
    int gridSize = (locationMessageCount + blockSize - 1) / blockSize;
    //Copy messages from d_messages to d_messages_swap, in hash order
    reorderLocationMessages <<<gridSize, blockSize, requiredSM_reorderLocationMessages(blockSize) >>>(d_keys, d_vals, d_PBM, d_locationMessages, d_locationMessages_swap);
    CUDA_CALL(hipDeviceSynchronize());//unncecssary sync
    swap();
    //Wait for return
    CUDA_CALL(hipDeviceSynchronize());
}
void SpatialPartition::swap()
{
    //Switch d_locationMessages and d_locationMessages_swap
    LocationMessages* d_locationmessages_temp = d_locationMessages;
    d_locationMessages = d_locationMessages_swap;
    d_locationMessages_swap = d_locationmessages_temp;

#ifdef _DEBUG
    PBM_isBuilt = 0;
    CUDA_CALL(hipMemcpyToSymbol(HIP_SYMBOL(d_PBM_isBuilt), &PBM_isBuilt, sizeof(unsigned int)));
#endif
}
void SpatialPartition::buildPBM()
{
    //If no messages, or instances, don't bother
    if (locationMessageCount<1) return;
#if _DEBUG
    assertSearch();
#endif
    //Fill primitive key/val arrays for sort
    launchHashLocationMessages();
    //Sort key val arrays using thrust/CUB
#ifndef THRUST
    //CUB version
    // Determine temporary device storage requirements
    void *d_temp_storage = NULL;
    size_t   temp_storage_bytes = 0;
    hipcub::DeviceRadixSort::SortPairs(d_temp_storage, temp_storage_bytes, d_keys, d_keys_swap, d_vals, d_vals_swap, locationMessageCount);
    // Allocate temporary storage
    hipMalloc(&d_temp_storage, temp_storage_bytes);
    // Run sorting operation
    hipcub::DeviceRadixSort::SortPairs(d_temp_storage, temp_storage_bytes, d_keys, d_keys_swap, d_vals, d_vals_swap, locationMessageCount);
    //Swap arrays
    unsigned int *temp;
    temp = d_keys;
    d_keys = d_keys_swap;
    d_keys_swap = temp;
    temp = d_vals;
    d_vals = d_vals_swap;
    d_vals_swap = temp;
    //Free temporary memory
    hipFree(d_temp_storage);
#else
    //Thrust version
    //hipStream_t s1;
    //hipStreamCreate(&s1);
    //thrust::sort_by_key(thrust::cuda::par(s1), d_keys, d_keys + locationMessageCount, d_vals);
    thrust::sort_by_key(thrust::cuda::par, d_keys, d_keys + locationMessageCount, d_vals);
    //hipStreamSynchronize(s1);
    //hipStreamDestroy(s1);
#endif
    CUDA_CALL(hipGetLastError());
    //Reorder map in order of message_hash	
    //Fill pbm start coords with known value 0xffffffff
    //CUDA_CALL(hipMemset(d_PBM, 0xffffffff, PARTITION_GRID_BIN_COUNT * sizeof(int)));
    //Fill pbm end coords with known value 0x00000000 (this should mean if the mysterious bug does occur, the cell is just dropped, not large loop created)
    unsigned int binCount = getBinCount(); 
    CUDA_CALL(hipMemset(d_PBM, 0x00000000, (binCount + 1) * sizeof(unsigned int)));
    launchReorderLocationMessages();

    //Clone data to textures ready for neighbourhood search
    fillTextures();
#ifdef _DEBUG
    PBM_isBuilt = 1;
    CUDA_CALL(hipMemcpyToSymbol(HIP_SYMBOL(d_PBM_isBuilt), &PBM_isBuilt, sizeof(unsigned int)));
#endif
}
