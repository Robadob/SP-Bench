#include "Neighbourhood.cuh"
#include "NeighbourhoodConstants.cuh"
#include "NeighbourhoodKernels.cuh"
#ifndef THRUST
#include <hipcub/hipcub.hpp>
#else
#include <thrust/sort.h>
#include <thrust/system/cuda/execution_policy.h>
#endif
#ifdef _GL
#include <cuda_gl_interop.h>
#endif
SpatialPartition::SpatialPartition(DIMENSIONS_VEC  environmentMin, DIMENSIONS_VEC environmentMax, unsigned int maxAgents, float interactionRad)
    : environmentMin(environmentMin)
    , environmentMax(environmentMax)
    , maxAgents(maxAgents)
    , interactionRad(interactionRad)
    , locationMessageCount(0)
    , gridDim((environmentMax - environmentMin) / interactionRad)
#ifdef _DEBUG
    , PBM_isBuilt(0)
#endif
{
    //Allocate bins in GPU memory
    deviceAllocateLocationMessages(&d_locationMessages);
    //Allocate bins swap in GPU memory
    deviceAllocateLocationMessages(&d_locationMessages_swap);
    //Allocate PBM
    deviceAllocatePBM(&d_PBM);
    //Allocate primitive structures
    deviceAllocatePrimitives(&d_keys, &d_vals);
#ifndef THRUST
    deviceAllocatePrimitives(&d_keys_swap, &d_vals_swap);
#endif
    //Allocate tex
    deviceAllocateTextures();
    //Set device constants
    CUDA_CALL(hipMemcpyToSymbol(HIP_SYMBOL(d_interactionRad), &interactionRad, sizeof(float)));
    CUDA_CALL(hipMemcpyToSymbol(HIP_SYMBOL(d_gridDim), &gridDim, sizeof(DIMENSIONS_IVEC)));
    DIMENSIONS_VEC t_gridDim = (DIMENSIONS_VEC)gridDim;
    CUDA_CALL(hipMemcpyToSymbol(HIP_SYMBOL(d_gridDim_float), &t_gridDim, sizeof(DIMENSIONS_VEC)));

    CUDA_CALL(hipMemcpyToSymbol(HIP_SYMBOL(d_environmentMin), &environmentMin, sizeof(DIMENSIONS_VEC)));
    CUDA_CALL(hipMemcpyToSymbol(HIP_SYMBOL(d_environmentMax), &environmentMax, sizeof(DIMENSIONS_VEC)));

#ifdef _DEBUG
    CUDA_CALL(hipMemcpyToSymbol(HIP_SYMBOL(d_PBM_isBuilt), &PBM_isBuilt, sizeof(unsigned int)));
#endif
    setLocationCount(locationMessageCount);
    unsigned int t_binCount = getBinCount();
    CUDA_CALL(hipMemcpyToSymbol(HIP_SYMBOL(d_binCount), &t_binCount, sizeof(unsigned int)));
}
SpatialPartition::~SpatialPartition()
{
    //Dellocate bins in GPU memory
    deviceDeallocateLocationMessages(d_locationMessages);
    //Dellocate bins swap in GPU memory
    deviceDeallocateLocationMessages(d_locationMessages_swap);
    //Dellocate PBM
    deviceDeallocatePBM(d_PBM);
    //Deallocated primitive structures
    deviceDeallocatePrimitives(d_keys, d_vals);
#ifndef THRUST
    deviceDeallocatePrimitives(d_keys_swap, d_vals_swap);
#endif
    //Deallocate tex
    deviceDeallocateTextures();
}
void SpatialPartition::deviceAllocateLocationMessages(LocationMessages **d_locMessage)
{
    unsigned int binCount = getBinCount();
    CUDA_CALL(hipMalloc(d_locMessage, sizeof(LocationMessages)));
    float *d_loc_temp;
    CUDA_CALL(hipMalloc(&d_loc_temp, sizeof(float)*maxAgents));
    CUDA_CALL(hipMemcpy(&((*d_locMessage)->locationX), &d_loc_temp, sizeof(float*), hipMemcpyHostToDevice));
    CUDA_CALL(hipMalloc(&d_loc_temp, sizeof(float)*maxAgents));
    CUDA_CALL(hipMemcpy(&((*d_locMessage)->locationY), &d_loc_temp, sizeof(float*), hipMemcpyHostToDevice));
#ifdef _3D
    CUDA_CALL(hipMalloc(&d_loc_temp, sizeof(float)*maxAgents));
    CUDA_CALL(hipMemcpy(&((*d_locMessage)->locationZ), &d_loc_temp, sizeof(float*), hipMemcpyHostToDevice));
#endif
#ifdef _GL
    CUDA_CALL(hipMalloc(&d_loc_temp, sizeof(float)*maxAgents));
    CUDA_CALL(hipMemcpy(&((*d_locMessage)->count), &d_loc_temp, sizeof(float*), hipMemcpyHostToDevice));
#endif
}
void SpatialPartition::deviceAllocatePBM(unsigned int **d_PBM_t)
{
    unsigned int binCount = getBinCount();
    CUDA_CALL(hipMalloc(d_PBM_t, sizeof(unsigned int)*(binCount+1)));
}
void SpatialPartition::deviceAllocatePrimitives(unsigned int **d_keys, unsigned int **d_vals)
{
    unsigned int binCount = getBinCount();
    CUDA_CALL(hipMalloc(d_keys, sizeof(unsigned int)*maxAgents));
    CUDA_CALL(hipMalloc(d_vals, sizeof(unsigned int)*maxAgents));
}
void SpatialPartition::deviceAllocateTextures()
{
    //Locations
#ifdef _GL
#pragma unroll 3
    for (unsigned int i = 0; i < DIMENSIONS;i++)
        deviceAllocateGLTexture_float(i);
    deviceAllocateGLTexture_float2();//Allocate a texture to store counting info in (Used to colour the visualisation
#else
#pragma unroll 3
    for (unsigned int i = 0; i < DIMENSIONS;i++)
        deviceAllocateTexture_float(i);
#endif
    //PBM
    deviceAllocateTexture_int();
}
void SpatialPartition::fillTextures()
{
    float *d_bufferPtr;
    //Potentially refactor so we store/swap these pointers on host in syncrhonisation
    CUDA_CALL(hipMemcpy(&d_bufferPtr, &d_locationMessages->locationX, sizeof(float*), hipMemcpyDeviceToHost));
    CUDA_CALL(hipMemcpy(tex_loc_ptr[0], d_bufferPtr, locationMessageCount*sizeof(float), hipMemcpyDeviceToDevice));
    CUDA_CALL(hipMemcpy(&d_bufferPtr, &d_locationMessages->locationY, sizeof(float*), hipMemcpyDeviceToHost));
    CUDA_CALL(hipMemcpy(tex_loc_ptr[1], d_bufferPtr, locationMessageCount*sizeof(float), hipMemcpyDeviceToDevice));
#ifdef _3D
    CUDA_CALL(hipMemcpy(&d_bufferPtr, &d_locationMessages->locationZ, sizeof(float*), hipMemcpyDeviceToHost));
    CUDA_CALL(hipMemcpy(tex_loc_ptr[2], d_bufferPtr, locationMessageCount*sizeof(float), hipMemcpyDeviceToDevice));
#endif
#ifdef _GL
    CUDA_CALL(hipMemcpy(&d_bufferPtr, &d_locationMessages->count, sizeof(float*), hipMemcpyDeviceToHost));
    CUDA_CALL(hipMemcpy(tex_location_ptr_count, d_bufferPtr, locationMessageCount*sizeof(float), hipMemcpyDeviceToDevice));
#endif

    CUDA_CALL(hipMemcpy(tex_PBM_ptr, d_PBM, (getBinCount()+1)*sizeof(unsigned int), hipMemcpyDeviceToDevice));
}

void SpatialPartition::deviceAllocateTexture_float(unsigned int i)
{
    if (i >= DIMENSIONS)
        return;
    //Allocate cuda array
    CUDA_CALL(hipMalloc(&tex_loc_ptr[i], maxAgents*sizeof(float)));
    //Define cuda resource from array
    hipResourceDesc resDesc;
    memset(&resDesc, 0, sizeof(hipResourceDesc));
    resDesc.resType = hipResourceTypeLinear;
    resDesc.res.linear.devPtr = tex_loc_ptr[i];
    resDesc.res.linear.desc.f = hipChannelFormatKindFloat;
    resDesc.res.linear.desc.x = 32; // bits per channel
    resDesc.res.linear.sizeInBytes = maxAgents*sizeof(float);
    //Define a cuda texture format
    hipTextureDesc texDesc;
    memset(&texDesc, 0, sizeof(hipTextureDesc));
    texDesc.readMode = hipReadModeElementType;
    //Create texture obj
    CUDA_CALL(hipCreateTextureObject(&tex_location[i], &resDesc, &texDesc, NULL));
    //Copy obj to const memory
    CUDA_CALL(hipMemcpyToSymbol(HIP_SYMBOL(d_tex_location), &tex_location[i], sizeof(hipTextureObject_t), i*sizeof(hipTextureObject_t)));
}
#ifdef _GL
void SpatialPartition::deviceAllocateGLTexture_float(unsigned int i)//GLuint *glTex, GLuint *glTbo, hipGraphicsResource_t *cuGres, hipArray_t *cuArr, hipTextureObject_t *tex, hipTextureObject_t *d_const, const unsigned int size)
{
    if (i >= DIMENSIONS)
        return;
    float *data = new float[maxAgents];
    //Gen tex
    GL_CALL(glGenTextures(1, &gl_tex[i]));
    //Gen buffer
    GL_CALL(glGenBuffers(1, &gl_tbo[i]));
    //Size buffer and tie to tex
    GL_CALL(glBindBuffer(GL_TEXTURE_BUFFER, gl_tbo[i]));
    GL_CALL(glBindBuffer(GL_TEXTURE_BUFFER, gl_tbo[i]));
    GL_CALL(glBufferData(GL_TEXTURE_BUFFER, maxAgents*sizeof(float), 0, GL_STATIC_DRAW));
   
    GL_CALL(glBindTexture(GL_TEXTURE_BUFFER, gl_tex[i]));
    //glBindTexture(GL_TEXTURE_2D, 0);
    GL_CALL(glTexBuffer(GL_TEXTURE_BUFFER, GL_R32F, gl_tbo[i]));
    GL_CALL(glBindBuffer(GL_TEXTURE_BUFFER, 0));
    GL_CALL(glBindTexture(GL_TEXTURE_BUFFER, 0));

    //Get CUDA handle to texture
    memset(&gl_gRes[i], 0, sizeof(hipGraphicsResource_t));
    CUDA_CALL(hipGraphicsGLRegisterBuffer(&gl_gRes[i], gl_tbo[i], cudaGraphicsMapFlagsNone));//GL_TEXTURE_BUFFER IS UNDOCUMENTED
    //Map/convert this to something hipArray_t
    CUDA_CALL(hipGraphicsMapResources(1, &gl_gRes[i]));
    CUDA_CALL(hipGraphicsResourceGetMappedPointer((void**)&tex_loc_ptr[i], 0, gl_gRes[i]));
    CUDA_CALL(hipGraphicsUnmapResources(1, &gl_gRes[i], 0));
    //Create a texture object from the hipArray_t
    hipResourceDesc resDesc;
    memset(&resDesc, 0, sizeof(hipResourceDesc));
    resDesc.resType = hipResourceTypeLinear;
    resDesc.res.linear.devPtr = tex_loc_ptr[i];
    resDesc.res.linear.desc.f = hipChannelFormatKindFloat;
    resDesc.res.linear.desc.x = 32; // bits per channel
    resDesc.res.linear.sizeInBytes = maxAgents*sizeof(float);
    hipTextureDesc texDesc;
    memset(&texDesc, 0, sizeof(hipTextureDesc));
    texDesc.readMode = hipReadModeElementType;
    CUDA_CALL(hipCreateTextureObject(&tex_location[i], &resDesc, &texDesc, nullptr));
    //Copy texture object to device constant
    CUDA_CALL(hipMemcpyToSymbol(HIP_SYMBOL(d_tex_location), &tex_location[i], sizeof(hipTextureObject_t), i*sizeof(hipTextureObject_t)));
    delete data;
}
#endif
/*
Allocates the PBM texture, which is only accessed via CUDA & memcpy
*/
void SpatialPartition::deviceAllocateTexture_int()
{
    //Define cuda array format
    //Allocate cuda array
    unsigned int size = getBinCount() + 1;
    CUDA_CALL(hipMalloc(&tex_PBM_ptr, size*sizeof(unsigned int)));//Number of elements, not bytes
    //Define cuda resource from array
    hipResourceDesc resDesc;
    memset(&resDesc, 0, sizeof(hipResourceDesc));
    resDesc.resType = hipResourceTypeLinear;
    resDesc.res.linear.devPtr = tex_PBM_ptr;
    resDesc.res.linear.desc.f = hipChannelFormatKindUnsigned;
    resDesc.res.linear.desc.x = 32; // bits per channel
    resDesc.res.linear.sizeInBytes = size*sizeof(unsigned int);

    hipTextureDesc texDesc;
    memset(&texDesc, 0, sizeof(hipTextureDesc));
    texDesc.readMode = hipReadModeElementType;

    CUDA_CALL(hipCreateTextureObject(&tex_PBM, &resDesc, &texDesc, NULL));
    CUDA_CALL(hipMemcpyToSymbol(HIP_SYMBOL(d_tex_PBM), &tex_PBM, sizeof(hipTextureObject_t)));
}
#ifdef _GL
/*
Allocates the count texture, which is only accessed via memcpy & GL
*/
void SpatialPartition::deviceAllocateGLTexture_float2()
{
    int *data = new int[maxAgents];
    //Gen tex
    GL_CALL(glGenTextures(1, &gl_tex_count));
    //Gen buffer
    GL_CALL(glGenBuffers(1, &gl_tbo_count));
    //Size buffer and tie to tex
    GL_CALL(glBindBuffer(GL_TEXTURE_BUFFER, gl_tbo_count));
    GL_CALL(glBindBuffer(GL_TEXTURE_BUFFER, gl_tbo_count));
    GL_CALL(glBufferData(GL_TEXTURE_BUFFER, maxAgents*sizeof(float), 0, GL_STATIC_DRAW));

    GL_CALL(glBindTexture(GL_TEXTURE_BUFFER, gl_tex_count));
    //glBindTexture(GL_TEXTURE_2D, 0);
    GL_CALL(glTexBuffer(GL_TEXTURE_BUFFER, GL_R32F, gl_tbo_count));
    GL_CALL(glBindBuffer(GL_TEXTURE_BUFFER, 0));
    GL_CALL(glBindTexture(GL_TEXTURE_BUFFER, 0));

    //Get CUDA handle to texture
    memset(&gl_gRes_count, 0, sizeof(hipGraphicsResource_t));
    CUDA_CALL(hipGraphicsGLRegisterBuffer(&gl_gRes_count, gl_tbo_count, cudaGraphicsMapFlagsNone));
    //Map/convert this to something hipArray_t
    CUDA_CALL(hipGraphicsMapResources(1, &gl_gRes_count));
    CUDA_CALL(hipGraphicsResourceGetMappedPointer((void**)&tex_location_ptr_count, 0, gl_gRes_count));
    CUDA_CALL(hipGraphicsUnmapResources(1, &gl_gRes_count, 0));
    delete data;
}
#endif
void SpatialPartition::deviceDeallocateLocationMessages(LocationMessages *d_locMessage)
{
    float *d_loc_temp;
    CUDA_CALL(hipMemcpy(&d_loc_temp, d_locMessage->locationX, sizeof(float*), hipMemcpyHostToDevice));
    CUDA_CALL(hipFree(d_loc_temp));
    CUDA_CALL(hipMemcpy(&d_loc_temp, d_locMessage->locationY, sizeof(float*), hipMemcpyHostToDevice));
    CUDA_CALL(hipFree(d_loc_temp));
#ifdef _3D
    CUDA_CALL(hipMemcpy(d_loc_temp, d_locMessage->locationZ, sizeof(float*), hipMemcpyHostToDevice));
    CUDA_CALL(hipFree(d_loc_temp));
#endif
    CUDA_CALL(hipFree(d_locMessage));
}
void SpatialPartition::deviceDeallocatePBM(unsigned int *d_PBM_t)
{
    CUDA_CALL(hipFree(d_PBM_t));
}
void SpatialPartition::deviceDeallocatePrimitives(unsigned int *d_keys, unsigned int *d_vals)
{
    CUDA_CALL(hipFree(d_keys));
    CUDA_CALL(hipFree(d_vals));
}
void SpatialPartition::deviceDeallocateTextures()
{

#pragma unroll
    for (unsigned int i = 0; i < DIMENSIONS; i++)
    {
        hipDestroyTextureObject(tex_location[i]);
#ifdef _GL
        hipGraphicsUnregisterResource(gl_gRes[i]);
        GL_CALL(glDeleteBuffers(1, &gl_tbo[i]));
        GL_CALL(glDeleteTextures(1, &gl_tex[i]));
#else
        hipFree(tex_loc_ptr[i]);
#endif
    }
    hipDestroyTextureObject(tex_PBM);
    hipFree(tex_PBM_ptr);
#ifdef _GL
    hipGraphicsUnregisterResource(gl_gRes_count);
    GL_CALL(glDeleteBuffers(1, &gl_tbo_count));
    GL_CALL(glDeleteTextures(1, &gl_tex_count));
#endif
}

unsigned int SpatialPartition::getBinCount()
{
    return (unsigned int)glm::compMul(gridDim);
}
void SpatialPartition::setLocationCount(unsigned int t_locationMessageCount)
{
    //Set local copy
    locationMessageCount = t_locationMessageCount;
    //Set device constants
    CUDA_CALL(hipMemcpyToSymbol(HIP_SYMBOL(d_locationMessageCount), &locationMessageCount, sizeof(unsigned int)));
}

void SpatialPartition::launchHashLocationMessages()
{
    int blockSize;   // The launch configurator returned block size 
    CUDA_CALL(hipOccupancyMaxActiveBlocksPerMultiprocessor(&blockSize, hashLocationMessages, 32, 0));//Randomly 32
    // Round up according to array size
    int gridSize = (locationMessageCount + blockSize - 1) / blockSize;
    hashLocationMessages <<<gridSize, blockSize>>>(d_keys, d_vals, d_locationMessages);
    CUDA_CALL(hipDeviceSynchronize());
}
int requiredSM_reorderLocationMessages(int blockSize)
{
    return sizeof(unsigned int)*blockSize;
}
void SpatialPartition::launchReorderLocationMessages()
{
    int minGridSize, blockSize;   // The launch configurator returned block size 
    hipOccupancyMaxPotentialBlockSizeVariableSMem(&minGridSize, &blockSize, reorderLocationMessages, requiredSM_reorderLocationMessages, 0);
    // Round up according to array size
    int gridSize = (locationMessageCount + blockSize - 1) / blockSize;
    //Copy messages from d_messages to d_messages_swap, in hash order
    reorderLocationMessages <<<gridSize, blockSize, requiredSM_reorderLocationMessages(blockSize) >>>(d_keys, d_vals, d_PBM, d_locationMessages, d_locationMessages_swap);
    CUDA_CALL(hipDeviceSynchronize());//unncecssary sync
    swap();
    //Wait for return
    CUDA_CALL(hipDeviceSynchronize());
}
void SpatialPartition::swap()
{
    //Switch d_locationMessages and d_locationMessages_swap
    LocationMessages* d_locationmessages_temp = d_locationMessages;
    d_locationMessages = d_locationMessages_swap;
    d_locationMessages_swap = d_locationmessages_temp;

#ifdef _DEBUG
    PBM_isBuilt = 0;
    CUDA_CALL(hipMemcpyToSymbol(HIP_SYMBOL(d_PBM_isBuilt), &PBM_isBuilt, sizeof(unsigned int)));
#endif
}
void SpatialPartition::buildPBM()
{
    //If no messages, or instances, don't bother
    if (locationMessageCount<1) return;
    //Fill primitive key/val arrays for sort
    launchHashLocationMessages();
    //Sort key val arrays using thrust/CUB
#ifndef THRUST
    //CUB version
    // Determine temporary device storage requirements
    void *d_temp_storage = NULL;
    size_t   temp_storage_bytes = 0;
    hipcub::DeviceRadixSort::SortPairs(d_temp_storage, temp_storage_bytes, d_keys, d_keys_swap, d_vals, d_vals_swap, locationMessageCount);
    // Allocate temporary storage
    hipMalloc(&d_temp_storage, temp_storage_bytes);
    // Run sorting operation
    hipcub::DeviceRadixSort::SortPairs(d_temp_storage, temp_storage_bytes, d_keys, d_keys_swap, d_vals, d_vals_swap, locationMessageCount);
    //Swap arrays
    unsigned int *temp;
    temp = d_keys;
    d_keys = d_keys_swap;
    d_keys_swap = temp;
    temp = d_vals;
    d_vals = d_vals_swap;
    d_vals_swap = temp;
    //Free temporary memory
    hipFree(d_temp_storage);
#else
    //Thrust version
    //hipStream_t s1;
    //hipStreamCreate(&s1);
    //thrust::sort_by_key(thrust::cuda::par(s1), d_keys, d_keys + locationMessageCount, d_vals);
    thrust::sort_by_key(thrust::cuda::par, d_keys, d_keys + locationMessageCount, d_vals);
    //hipStreamSynchronize(s1);
    //hipStreamDestroy(s1);
#endif
    CUDA_CALL(hipGetLastError());
    //Reorder map in order of message_hash	
    //Fill pbm start coords with known value 0xffffffff
    //CUDA_CALL(hipMemset(d_PBM, 0xffffffff, PARTITION_GRID_BIN_COUNT * sizeof(int)));
    //Fill pbm end coords with known value 0x00000000 (this should mean if the mysterious bug does occur, the cell is just dropped, not large loop created)
    unsigned int binCount = getBinCount(); 
    CUDA_CALL(hipMemset(d_PBM, 0x00000000, (binCount + 1) * sizeof(unsigned int)));
    launchReorderLocationMessages();

    //Clone data to textures ready for neighbourhood search
    fillTextures();
#ifdef _DEBUG
    PBM_isBuilt = 1;
    CUDA_CALL(hipMemcpyToSymbol(HIP_SYMBOL(d_PBM_isBuilt), &PBM_isBuilt, sizeof(unsigned int)));
#endif
}
