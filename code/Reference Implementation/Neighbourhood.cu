#include "hip/hip_runtime.h"
#include "Neighbourhood.cuh"
#include "NeighbourhoodKernels.cuh"
#include "NeighbourhoodConstants.cuh"
#ifndef THRUST
#include <cub\cub.cuh>
#else
#include <thrust\sort.h>
#include <thrust/system/cuda/execution_policy.h>
#endif

#ifdef _3D
SpatialPartition::SpatialPartition(glm::vec3  environmentMin, glm::vec3 environmentMax, unsigned int maxAgents, float neighbourRad)
#else
SpatialPartition::SpatialPartition(glm::vec2  environmentMin, glm::vec2 environmentMax, unsigned int maxAgents, float neighbourRad)
#endif
    : environmentMin(environmentMin)
    , environmentMax(environmentMax)
    , maxAgents(maxAgents)
    , neighbourRad(neighbourRad)
    , locationMessageCount(0)
    , gridDim((environmentMax - environmentMin) / neighbourRad)
{
    //Allocate bins in GPU memory
    deviceAllocateLocationMessages(&d_locationMessages);
    //Allocate bins swap in GPU memory
    deviceAllocateLocationMessages(&d_locationMessages_swap);
    //Allocate PBM
    deviceAllocatePBM(&d_PBM);
    //Allocate primitive structures
    deviceAllocatePrimitives(&d_keys, &d_vals);
#ifndef THRUST
    deviceAllocatePrimitives(&d_keys_swap, &d_vals_swap);
#endif
    //Set device constants
#ifdef _3D
    hipMemcpyToSymbol(HIP_SYMBOL(&d_gridDim), &gridDim, sizeof(glm::ivec3));
    hipMemcpyToSymbol(HIP_SYMBOL(&d_environmentMin), &environmentMin, sizeof(glm::vec3));
    hipMemcpyToSymbol(HIP_SYMBOL(&d_environmentMax), &environmentMax, sizeof(glm::vec3));
#else
    hipMemcpyToSymbol(HIP_SYMBOL(&d_gridDim), &gridDim, sizeof(glm::ivec2));
    hipMemcpyToSymbol(HIP_SYMBOL(&d_environmentMin), &environmentMin, sizeof(glm::vec2));
    hipMemcpyToSymbol(HIP_SYMBOL(&d_environmentMax), &environmentMax, sizeof(glm::vec2));
#endif
    setLocationCount(locationMessageCount);
}
SpatialPartition::~SpatialPartition()
{
    //Dellocate bins in GPU memory
    deviceDeallocateLocationMessages(d_locationMessages);
    //Dellocate bins swap in GPU memory
    deviceDeallocateLocationMessages(d_locationMessages_swap);
    //Dellocate PBM
    deviceDeallocatePBM(d_PBM);
    //Deallocated primitive structures
    deviceDeallocatePrimitives(d_keys, d_vals);
#ifndef THRUST
    deviceDeallocatePrimitives(d_keys_swap, d_vals_swap);
#endif
}
void SpatialPartition::deviceAllocateLocationMessages(LocationMessages **d_locMessage)
{
    unsigned int binCount = getBinCount();
    CUDA_CALL(hipMalloc(d_locMessage, sizeof(LocationMessages)));
    float *d_loc_temp;
    CUDA_CALL(hipMalloc(&d_loc_temp, sizeof(float)*binCount));
    CUDA_CALL(hipMemcpy((*d_locMessage)->locationX, d_loc_temp, sizeof(float*), hipMemcpyHostToDevice));
    CUDA_CALL(hipMalloc(&d_loc_temp, sizeof(float)*binCount));
    CUDA_CALL(hipMemcpy((*d_locMessage)->locationY, d_loc_temp, sizeof(float*), hipMemcpyHostToDevice));
#ifdef _3D
    CUDA_CALL(hipMalloc(&d_loc_temp, sizeof(float)*binCount));
    CUDA_CALL(hipMemcpy((*d_locMessage)->locationZ, d_loc_temp, sizeof(float*), hipMemcpyHostToDevice));
#endif
}
void SpatialPartition::deviceAllocatePBM(unsigned int **d_PBM_t)
{
    unsigned int binCount = getBinCount();
    CUDA_CALL(hipMalloc(d_PBM_t, sizeof(unsigned int)*binCount));
}
void SpatialPartition::deviceAllocatePrimitives(unsigned int **d_keys, unsigned int **d_vals)
{
    unsigned int binCount = getBinCount();
    CUDA_CALL(hipMalloc(d_keys, sizeof(unsigned int)*binCount));
    CUDA_CALL(hipMalloc(d_vals, sizeof(unsigned int)*binCount));
}
void SpatialPartition::deviceAllocateTextures()
{
    float *d_bufferPtr;
    //Potentially refactor so we store/swap these pointers on host in syncrhonisation
    CUDA_CALL(hipMemcpy(&d_bufferPtr, d_locationMessages->locationX, sizeof(float*), hipMemcpyDeviceToHost));
    deviceAllocateTexture_float(&tex_locationX, d_bufferPtr, locationMessageCount);
    CUDA_CALL(hipMemcpy(&d_bufferPtr, d_locationMessages->locationY, sizeof(float*), hipMemcpyDeviceToHost));
    deviceAllocateTexture_float(&tex_locationY, d_bufferPtr, locationMessageCount);
#ifdef _3D
    CUDA_CALL(hipMemcpy(&d_bufferPtr, d_locationMessages->locationZ, sizeof(float*), hipMemcpyDeviceToHost));
    deviceAllocateTexture_float(&tex_locationZ, d_bufferPtr, locationMessageCount);
#endif
    //PBM
    deviceAllocateTexture_int(&tex_PBM, d_PBM, getBinCount());
}

void SpatialPartition::deviceAllocateTexture_float(hipTextureObject_t *tex, float* d_data, const int size)
{
    hipResourceDesc resDesc;
    memset(&resDesc, 0, sizeof(hipResourceDesc));
    resDesc.resType = hipResourceTypeLinear;
    resDesc.res.linear.devPtr = d_data;
    resDesc.res.linear.desc.f = hipChannelFormatKindFloat;
    resDesc.res.linear.desc.x = 32; // bits per channel
    resDesc.res.linear.sizeInBytes = size*sizeof(float);

    hipTextureDesc texDesc;
    memset(&texDesc, 0, sizeof(hipTextureDesc));
    texDesc.readMode = hipReadModeElementType;

    hipCreateTextureObject(tex, &resDesc, &texDesc, NULL);
}
void SpatialPartition::deviceAllocateTexture_int(hipTextureObject_t *tex, unsigned int* d_data, const int size)
{
    hipResourceDesc resDesc;
    memset(&resDesc, 0, sizeof(hipResourceDesc));
    resDesc.resType = hipResourceTypeLinear;
    resDesc.res.linear.devPtr = d_data;
    resDesc.res.linear.desc.f = hipChannelFormatKindUnsigned;
    resDesc.res.linear.desc.x = 32; // bits per channel
    resDesc.res.linear.sizeInBytes = size*sizeof(unsigned int);

    hipTextureDesc texDesc;
    memset(&texDesc, 0, sizeof(hipTextureDesc));
    texDesc.readMode = hipReadModeElementType;

    hipCreateTextureObject(tex, &resDesc, &texDesc, NULL);
}
void SpatialPartition::deviceDeallocateLocationMessages(LocationMessages *d_locMessage)
{
    float *d_loc_temp;
    CUDA_CALL(hipMemcpy(&d_loc_temp, d_locMessage->locationX, sizeof(float*), hipMemcpyHostToDevice));
    CUDA_CALL(hipFree(d_loc_temp));
    CUDA_CALL(hipMemcpy(&d_loc_temp, d_locMessage->locationY, sizeof(float*), hipMemcpyHostToDevice));
    CUDA_CALL(hipFree(d_loc_temp));
#ifdef _3D
    CUDA_CALL(hipMemcpy(d_loc_temp, d_locMessage->locationZ, sizeof(float*), hipMemcpyHostToDevice));
    CUDA_CALL(hipFree(d_loc_temp));
#endif
    CUDA_CALL(hipFree(d_locMessage));
}
void SpatialPartition::deviceDeallocatePBM(unsigned int *d_PBM_t)
{
    CUDA_CALL(hipFree(d_PBM_t));
}
void SpatialPartition::deviceDeallocatePrimitives(unsigned int *d_keys, unsigned int *d_vals)
{
    CUDA_CALL(hipFree(d_keys));
    CUDA_CALL(hipFree(d_vals));
}
void SpatialPartition::deviceDeallocateTextures()
{
    hipDestroyTextureObject(tex_locationX);
    hipDestroyTextureObject(tex_locationY);
#ifdef _3D
    hipDestroyTextureObject(tex_locationZ);
#endif
    hipDestroyTextureObject(tex_PBM);
}

unsigned int SpatialPartition::getBinCount()
{
    return (unsigned int)glm::compMul((environmentMax - environmentMin) / neighbourRad);
}
void SpatialPartition::setLocationCount(unsigned int t_locationMessageCount)
{
    //Set local copy
    locationMessageCount = t_locationMessageCount;
    //Set device constants
    hipMemcpyToSymbol(HIP_SYMBOL(&d_locationMessageCount), &locationMessageCount, sizeof(unsigned int));
}

void SpatialPartition::launchHashLocationMessages()
{
    int blockSize;   // The launch configurator returned block size 
    hipOccupancyMaxActiveBlocksPerMultiprocessor(&blockSize, hashLocationMessages, 0, 0);
    // Round up according to array size
    int gridSize = (locationMessageCount + blockSize - 1) / blockSize;
    hashLocationMessages <<<gridSize, blockSize>>>(d_keys, d_vals, d_locationMessages);
    hipDeviceSynchronize();
    CUDA_CALL(hipGetLastError());
}
int requiredSM_reorderLocationMessages(int blockSize)
{
    return sizeof(unsigned int)*blockSize;
}
void SpatialPartition::launchReorderLocationMessages()
{
    int minGridSize, blockSize;   // The launch configurator returned block size 
    hipOccupancyMaxPotentialBlockSizeVariableSMem(&minGridSize, &blockSize, reorderLocationMessages, requiredSM_reorderLocationMessages, 0);
    // Round up according to array size
    int gridSize = (locationMessageCount + blockSize - 1) / blockSize;
    //Copy messages from d_messages to d_messages_swap, in hash order
    reorderLocationMessages <<<gridSize, blockSize, requiredSM_reorderLocationMessages(blockSize) >>>(d_keys, d_vals, d_PBM, d_locationMessages, d_locationMessages_swap);
    //Switch d_locationMessages and d_locationMessages_swap
    LocationMessages* d_locationmessages_temp = d_locationMessages;
    d_locationMessages = d_locationMessages_swap;
    d_locationMessages_swap = d_locationmessages_temp;
    //Wait for return
    hipDeviceSynchronize();
    CUDA_CALL(hipGetLastError());
}
void SpatialPartition::buildPBM()
{
    //Clear previous textures
    deviceDeallocateTextures();

    //If no messages, or instances, don't bother
    if (locationMessageCount<1) return;
    //Fill primitive key/val arrays for sort
    launchHashLocationMessages();
    //Sort key val arrays using thrust/CUB
#ifndef THRUST
    //CUB version
    // Determine temporary device storage requirements
    void *d_temp_storage = NULL;
    size_t   temp_storage_bytes = 0;
    hipcub::DeviceRadixSort::SortPairs(d_temp_storage, temp_storage_bytes, d_keys, d_keys_swap, d_vals, d_vals_swap, locationMessageCount);
    // Allocate temporary storage
    hipMalloc(&d_temp_storage, temp_storage_bytes);
    // Run sorting operation
    hipcub::DeviceRadixSort::SortPairs(d_temp_storage, temp_storage_bytes, d_keys, d_keys_swap, d_vals, d_vals_swap, locationMessageCount);
    //Swap arrays
    unsigned int *temp;
    temp = d_keys;
    d_keys = d_keys_swap;
    d_keys_swap = temp;
    temp = d_vals;
    d_vals = d_vals_swap;
    d_vals_swap = temp;
    //Free temporary memory
    hipFree(d_temp_storage);

    //Clone data to textures ready for neighbourhood search
    deviceAllocateTextures();
#else
    //Thrust version
    //hipStream_t s1;
    //hipStreamCreate(&s1);
    //thrust::sort_by_key(thrust::cuda::par(s1), d_keys, d_keys + locationMessageCount, d_vals);
    thrust::sort_by_key(thrust::cuda::par, d_keys, d_keys + locationMessageCount, d_vals);
    //hipStreamSynchronize(s1);
    //hipStreamDestroy(s1);
#endif
    CUDA_CALL(hipGetLastError());
    //Reorder map in order of message_hash	
    //Fill pbm start coords with known value 0xffffffff
    //CUDA_CALL(hipMemset(d_PBM, 0xffffffff, PARTITION_GRID_BIN_COUNT * sizeof(int)));
    //Fill pbm end coords with known value 0x00000000 (this should mean if the mysterious bug does occur, the cell is just dropped, not large loop created)
    unsigned int binCount = getBinCount(); 
    CUDA_CALL(hipMemset(d_PBM, 0x00000000, binCount * sizeof(unsigned int)));
    launchReorderLocationMessages();
}










//
//
//
//
////#include "CudppMgr.h"//Replacing cudpp with CUB
//#include <limits.h>
//#include <float.h>
////#include "Environment.h"
/////Device constants 
//namespace Neighbourhood
//{
//    __constant__ int COUNT__;
//    __constant__ int MESSAGE_COUNT__;
//    __constant__ int3 GRID_DIMENSIONS__;
//    __constant__ float SEARCH_RADIUS__;
//    //Global Textures & Texture Constants
//    __constant__ int d_tex_message_location_x_offset;
//    __constant__ int d_tex_message_location_y_offset;
//    __constant__ int d_tex_message_location_z_offset;
//    __constant__ int d_tex_message_radius_offset;
//    __constant__ int d_tex_message_ent_id_offset;
//    __constant__ int d_tex_message_team_offset;
//    __constant__ int d_tex_pbm_start_offset;
//    __constant__ int d_tex_pbm_end_offset;
//    //Texture binding vars (used for reading location messages more efficiently that from global memory)
//    //These aren't valid c++ prior to being parsed by nvcc, so must live in a .cu file
//    //1 per LocationMessage part
//    texture<float, 1, hipReadModeElementType> tex_message_location_x;
//    texture<float, 1, hipReadModeElementType> tex_message_location_y;
//#ifdef 3D
//    texture<float, 1, hipReadModeElementType> tex_message_location_z;
//#endif
//    ////Pbm ones
//    texture<int, 1, hipReadModeElementType> tex_pbm_start;
//
//    __device__ bool loadNextLocationMessage(int3 relative_bin, uint bin_index_max, int3 central_bin, int bin_index)
//    {
//        extern __shared__ int sm_data[];
//        char* message_share = (char*)&sm_data[0];
//
//        int change_bin = true;
//        bin_index++;
//
//        //Check if there are messages left to check in current bin
//        if (bin_index < bin_index_max)
//            change_bin = false;
//
//        while (change_bin)
//        {
//            //get the next relative grid position 
//            if (getNextBin(&relative_bin))
//            {
//                //calculate the next cells grid position and hash
//                int3 next_bin_position = central_bin + relative_bin;
//                int next_bin_hash = getHash(next_bin_position);
//                //use the hash to calculate the start index
//                uint bin_index_min = tex1Dfetch(tex_pbm_start, next_bin_hash + d_tex_pbm_start_offset);
//
//                //check for messages in the cell (empty cells with have a start index of 0xffffffff)
//                if (bin_index_min != 0xffffffff)
//                {
//                    //if there are messages in the cell then update the cell index max value
//                    bin_index_max = tex1Dfetch(tex_pbm_end, next_bin_hash + d_tex_pbm_end_offset);
//                    //start from the cell index min
//                    bin_index = bin_index_min;
//                    //exit the loop as we have found a valid cell with message data
//                    change_bin = false;
//                }
//            }
//            else
//            {
//                //We have exhausted all the neightbouring cells so there are no more messages
//                return false;
//            }
//        }
//
//        LocationMessage temp_message;
//
//        //get message data using texture fetch
//        temp_message._relative_bin = relative_bin;
//        temp_message._bin_index_max = bin_index_max;
//        temp_message._bin_index = bin_index;
//        temp_message._central_bin = central_bin;
//
//        //Using texture cache
//        temp_message.position.x = tex1Dfetch(tex_message_location_x, bin_index + d_tex_message_location_x_offset);
//        temp_message.position.y = tex1Dfetch(tex_message_location_y, bin_index + d_tex_message_location_y_offset);
//        temp_message.position.z = tex1Dfetch(tex_message_location_z, bin_index + d_tex_message_location_z_offset);
//        temp_message.bounding_radius = tex1Dfetch(tex_message_radius, bin_index + d_tex_message_radius_offset);
//        temp_message.ent_id = tex1Dfetch(tex_message_ent_id, bin_index + d_tex_message_ent_id_offset);
//        temp_message.team = tex1Dfetch(tex_message_team, bin_index + d_tex_message_team_offset);
//
//        //load it into shared memory (no sync as no sharing between threads)
//        int message_index = threadIdx.x * sizeof(LocationMessage);
//        LocationMessage* sm_message = ((LocationMessage*)&message_share[message_index]);
//        sm_message[0] = temp_message;
//        return true;
//    }
//    void alloc()
//    {//Only run first time, to malloc statics
//        h_messageCount = 12;
//        CUDA_CALL(hipMemcpyToSymbol(HIP_SYMBOL(MESSAGE_COUNT__), &h_messageCount, sizeof(int)));
//        //Malloc
//        CUDA_CALL(hipMalloc(&d_messages, sizeof(LocationMessageList)));
//        CUDA_CALL(hipMalloc(&d_messages_swap, sizeof(LocationMessageList)));
//        CUDA_CALL(hipMalloc(&d_PBM, sizeof(PartitionBoundaryMatrix)));
//        //ccTex = new CCTextures;
//        //CUDA Constants
//        float t_searchRad = SEARCH_RADIUS;
//        CUDA_CALL(hipMemcpyToSymbol(HIP_SYMBOL(SEARCH_RADIUS__), &t_searchRad, sizeof(float)));
//        int3 t_paritionDims = make_int3(
//            (int)ceil((WORLD_X_MAX - WORLD_X_MIN) / (float)SEARCH_RADIUS),
//            (int)ceil((WORLD_Y_MAX - WORLD_Y_MIN) / (float)SEARCH_RADIUS),
//            (int)ceil((WORLD_Z_MAX - WORLD_Z_MIN) / (float)SEARCH_RADIUS)
//            );
//        CUDA_CALL(hipMemcpyToSymbol(HIP_SYMBOL(GRID_DIMENSIONS__), &t_paritionDims, sizeof(int3)));
//    }
//    void free()
//    {
//        hipFree(&d_messages);
//        hipFree(&d_messages_swap);
//        hipFree(&d_PBM);
//    }
//    void clearBuffer()
//    {
//        //If no messages, or instances, don't bother
//        if (h_messageCount<1) return;
//        h_messageCount = 0;
//        CUDA_CALL(hipMemcpyToSymbol(HIP_SYMBOL(MESSAGE_COUNT__), &h_messageCount, sizeof(int)));
//    }
//    void buildPBM()
//    {
//
//    }
//    PartitionBoundaryMatrix *allocCollisionBuffers()
//    {
//        ///Bind location message buffer to textures
//        //LocationMessage->position.x
//        CUDA_CALL(hipGetLastError());
//        size_t tex_message_x_byte_offset;
//        hipChannelFormatDesc tex_desc_x = hipCreateChannelDesc<float>();
//        CUDA_CALL(hipBindTexture(&tex_message_x_byte_offset, &tex_message_location_x, d_messages->position_x, &tex_desc_x, sizeof(float)*LOCATION_MESSAGE_MAX));//sizeof(int)
//        int h_tex_message_location_x_offset = (int)tex_message_x_byte_offset / sizeof(float);
//        CUDA_CALL(hipMemcpyToSymbol(HIP_SYMBOL(d_tex_message_location_x_offset), &h_tex_message_location_x_offset, sizeof(int)));
//        //LocationMessage->position.y
//        size_t tex_message_y_byte_offset;
//        hipChannelFormatDesc tex_desc_y = hipCreateChannelDesc<float>();
//        CUDA_CALL(hipBindTexture(&tex_message_y_byte_offset, &tex_message_location_y, d_messages->position_y, &tex_desc_y, sizeof(float)*LOCATION_MESSAGE_MAX));//sizeof(int)
//        int h_tex_message_location_y_offset = (int)tex_message_y_byte_offset / sizeof(float);
//        CUDA_CALL(hipMemcpyToSymbol(HIP_SYMBOL(d_tex_message_location_y_offset), &h_tex_message_location_y_offset, sizeof(int)));
//        //LocationMessage->position.z
//        size_t tex_message_z_byte_offset;
//        hipChannelFormatDesc tex_desc_z = hipCreateChannelDesc<float>();
//        CUDA_CALL(hipBindTexture(&tex_message_z_byte_offset, &tex_message_location_z, d_messages->position_z, &tex_desc_z, sizeof(float)*LOCATION_MESSAGE_MAX));//sizeof(int)
//        int h_tex_message_location_z_offset = (int)tex_message_z_byte_offset / sizeof(float);
//        CUDA_CALL(hipMemcpyToSymbol(HIP_SYMBOL(d_tex_message_location_z_offset), &h_tex_message_location_z_offset, sizeof(int)));
//        //LocationMessage->bounding_radius
//        size_t d_tex_message_radius_byte_offset;
//        hipChannelFormatDesc tex_desc_radius = hipCreateChannelDesc<float>();
//        CUDA_CALL(hipBindTexture(&d_tex_message_radius_byte_offset, &tex_message_radius, d_messages->bounding_radius, &tex_desc_radius, sizeof(float)*LOCATION_MESSAGE_MAX));//sizeof(int)
//        int h_tex_message_radius_offset = (int)d_tex_message_radius_byte_offset / sizeof(float);
//        CUDA_CALL(hipMemcpyToSymbol(HIP_SYMBOL(d_tex_message_location_y_offset), &h_tex_message_radius_offset, sizeof(int)));
//        size_t d_tex_message_ent_id_byte_offset;
//        hipChannelFormatDesc tex_desc_ent_id = hipCreateChannelDesc<float>();
//        CUDA_CALL(hipBindTexture(&d_tex_message_ent_id_byte_offset, &tex_message_ent_id, d_messages->ent_id, &tex_desc_ent_id, sizeof(float)*LOCATION_MESSAGE_MAX));//sizeof(int)
//        int h_tex_message_ent_id_offset = (int)d_tex_message_ent_id_byte_offset / sizeof(float);
//        CUDA_CALL(hipMemcpyToSymbol(HIP_SYMBOL(d_tex_message_location_y_offset), &h_tex_message_ent_id_offset, sizeof(int)));
//        //LocationMessage->team
//        size_t d_tex_message_team_byte_offset;
//        hipChannelFormatDesc tex_desc_team = hipCreateChannelDesc<float>();
//        CUDA_CALL(hipBindTexture(&d_tex_message_team_byte_offset, &tex_message_team, d_messages->team, &tex_desc_team, sizeof(float)*LOCATION_MESSAGE_MAX));//sizeof(int)
//        int h_tex_message_team_offset = (int)d_tex_message_team_byte_offset / sizeof(float);
//        CUDA_CALL(hipMemcpyToSymbol(HIP_SYMBOL(d_tex_message_location_y_offset), &h_tex_message_team_offset, sizeof(int)));
//
//        ///Bind PBM start and end indices to textures
//        //Start
//        size_t d_tex_pbm_start_byte_offset;
//        hipChannelFormatDesc tex_desc_pbm_start = hipCreateChannelDesc<int>();
//        CUDA_CALL(hipBindTexture(&d_tex_pbm_start_byte_offset, &tex_pbm_start, d_PBM->start, &tex_desc_pbm_start, sizeof(int)*PARTITION_GRID_BIN_COUNT));
//        int h_tex_pbm_start_offset = (int)d_tex_pbm_start_byte_offset / sizeof(int);
//        CUDA_CALL(hipMemcpyToSymbol(HIP_SYMBOL(d_tex_pbm_start_offset), &h_tex_pbm_start_offset, sizeof(int)));
//        //End
//        size_t d_tex_pbm_end_byte_offset;
//        hipChannelFormatDesc tex_desc_pbm_end = hipCreateChannelDesc<int>();
//        CUDA_CALL(hipBindTexture(&d_tex_pbm_end_byte_offset, &tex_pbm_end, d_PBM->end, &tex_desc_pbm_end, sizeof(int)*PARTITION_GRID_BIN_COUNT));
//        int h_tex_pbm_end_offset = (int)d_tex_pbm_end_byte_offset / sizeof(int);
//        CUDA_CALL(hipMemcpyToSymbol(HIP_SYMBOL(d_tex_pbm_end_offset), &h_tex_pbm_end_offset, sizeof(int)));
//
//        return d_PBM;
//    }
//    void freeCollisionBuffers()
//    {
//        CUDA_CALL(hipUnbindTexture(tex_message_location_x));
//        CUDA_CALL(hipUnbindTexture(tex_message_location_y));
//        CUDA_CALL(hipUnbindTexture(tex_message_location_z));
//        CUDA_CALL(hipUnbindTexture(tex_message_radius));
//        CUDA_CALL(hipUnbindTexture(tex_message_ent_id));
//        CUDA_CALL(hipUnbindTexture(tex_message_team));
//        CUDA_CALL(hipUnbindTexture(tex_pbm_start));
//        CUDA_CALL(hipUnbindTexture(tex_pbm_end));
//    }
//    __device__ LocationMessage *getFirstLocationMessage(float x, float y, float z)
//    {
//        extern __shared__ int sm_data[];
//        char* message_share = (char*)&sm_data[0];
//
//        int3 relative_bin = make_int3(-2, -1, -1);//Start out of range, so we get moved into 1st cell
//        int bin_index_max = 0;
//        int bin_index = 0;
//        float3 position = make_float3(x, y, z);
//        int3 agent_grid_bin = getGridPosition(position);
//
//        if (loadNextLocationMessage(relative_bin, bin_index_max, agent_grid_bin, bin_index))
//        {
//            int message_index = __mul24(threadIdx.x, sizeof(LocationMessage));
//            return ((LocationMessage*)&message_share[message_index]);
//        }
//        else
//        {
//            return 0;
//        }
//    }
//    __device__ LocationMessage *getFirstLocationMessage(float3 position)
//    {
//        extern __shared__ int sm_data[];
//        char* message_share = (char*)&sm_data[0];
//
//        int3 relative_bin = make_int3(-2, -1, -1);//Start out of range, so we get moved into 1st cell
//        int bin_index_max = 0;
//        int bin_index = 0;
//        int3 agent_grid_cell = getGridPosition(position);
//
//        if (loadNextLocationMessage(relative_bin, bin_index_max, agent_grid_cell, bin_index))
//        {
//            int message_index = __mul24(threadIdx.x, sizeof(LocationMessage));
//            return ((LocationMessage*)&message_share[message_index]);
//        }
//        else
//        {
//            return 0;
//        }
//    }
//    __device__ LocationMessage *getNextLocationMessage(LocationMessage *message)
//    {
//        extern __shared__ int sm_data[];
//        char* message_share = (char*)&sm_data[0];
//
//        if (loadNextLocationMessage(message->_relative_bin, message->_bin_index_max, message->_central_bin, message->_bin_index))
//        {
//            int message_index = __mul24(threadIdx.x, sizeof(LocationMessage));
//            return ((LocationMessage*)&message_share[message_index]);
//        }
//        else
//        {
//            return 0;
//        }
//    }
//
//    __constant__ int ID_OFF__;
//    int *getMessageCount(){ return &h_messageCount; }
//    template <class T_EntityList>
//    __global__ void addEntitySetToBuffer(T_EntityList *entity_buffer, CollisionCore::LocationMessageList *message_buffer)
//    {
//        int index = __mul24(blockIdx.x, blockDim.x) + threadIdx.x;
//        //Agent out of bounds
//        if (index >= COUNT__) return;
//        int message_index = index + MESSAGE_COUNT__;
//        message_buffer->position_x[message_index] = entity_buffer->position_x[index];
//        message_buffer->position_y[message_index] = entity_buffer->position_y[index];
//        message_buffer->position_z[message_index] = entity_buffer->position_z[index];
//        message_buffer->bounding_radius[message_index] = entity_buffer->bounding_radius[index];
//        message_buffer->team[message_index] = GET_TEAM_MASK(entity_buffer->states[index]);
//        message_buffer->ent_id[message_index] = entity_buffer->_id[index] + ID_OFF__;
//    }
//
//    template <class T_EntityList>
//    void postLocationMessages(T_EntityList *d_entities, int count, int id_offset)
//    {
//        if (!count)
//            return;
//        if (h_messageCount + count > LOCATION_MESSAGE_MAX){
//            printf("Error: Location message buffer would be exceeded, skipping addition of entities.\n");
//            return;
//        }
//        CUDA_CALL(hipMemcpyToSymbol(HIP_SYMBOL(COUNT__), &count, sizeof(int)));
//        CUDA_CALL(hipMemcpyToSymbol(HIP_SYMBOL(ID_OFF__), &id_offset, sizeof(int)));
//        KERNEL_PARAMS(count)
//            addEntitySetToBuffer << <grid, threads >> >(d_entities, d_messages);
//        h_messageCount += count;
//        hipDeviceSynchronize();//Don't update device msgCount const until kernal returns
//        CUDA_CALL(hipGetLastError());
//        CollisionCore::updateDMessageCount();
//    }
//    namespace
//    {
//        __device__ int getNextBin(int3* relative_bin)
//        {
//            int3 oldbin = make_int3(relative_bin->x, relative_bin->y, relative_bin->z);
//            if (relative_bin->x < 1)
//            {
//                relative_bin->x++;
//                return true;
//            }
//            relative_bin->x = -1;
//
//            if (relative_bin->y < 1)
//            {
//                relative_bin->y++;
//                return true;
//            }
//            relative_bin->y = -1;
//
//            if (relative_bin->z < 1)
//            {
//                relative_bin->z++;
//                return true;
//            }
//            relative_bin->z = -1;
//            return false;
//        }
//    }
//}

#define __Neighbourhood_cuh__
//#include "Asteroid.h"//Using C++ templates properly should remove the requirement of this
#undef __Neighbourhood_cuh__